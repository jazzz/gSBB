#include "hip/hip_runtime.h"
#include "CudaControllerFunc.cuh"
#include "CudaControllerVars.cuh"

#define _DEVICE_VARS_

#define WARP_SIZE 32
#define MODE_MASK 0x1
#define OP_MASK 0xE
#define DST_MASK 0x70
#define SRC_MASK 0x1F80

#define MODE_SHIFT 0
#define OP_SHIFT 1
#define DST_SHIFT 4
#define SRC_SHIFT 7

#define REG_COUNT 8

#define OP_SUM 0
#define OP_DIFF 1
#define OP_PROD 2
#define OP_DIV 3
#define OP_MOD 4
#define OP_COS 5
#define OP_EXP 6
#define OP_LOG 7
#define OP_CODE_COUNT 8


#define OpCodeCount 7
#define OP_NO 199
#define PIVOT_STRIPE_SIZE 16

__constant__ short NUM_FEATURES;
__constant__ short LEARNER_LENGTH;
__constant__ int TOTAL_LEARNERS;
__constant__ int TOTAL_POINTS;


__global__ void kLearnerEval(int learnOffset,
                      int pointOffset,
                      _learner* dLearnerMatrix,
                      _learnerBid* dLearnerBidMatrix,
                      _point* dPointMatrix,
                      int learnerCount,
                      int pointCount)
{

    int pointId = threadIdx.x + blockIdx.x * blockDim.x + pointOffset; // + (blockIdx.x*gridDim.x);
    int learnerId = threadIdx.y + blockIdx.y* blockDim.y + learnOffset;// + blockIdx.y * blockDim.y;
    if (learnerId < TOTAL_LEARNERS && pointId < TOTAL_POINTS)
    {
		int id = threadIdx.x*blockDim.y + threadIdx.y;
		_learner *learner = &dLearnerMatrix[(threadIdx.y + blockIdx.y*blockDim.y) * LEARNER_LENGTH];
        _point *feature = &dPointMatrix[(threadIdx.x+blockDim.x*blockIdx.x)/PIVOT_STRIPE_SIZE * PIVOT_STRIPE_SIZE* NUM_FEATURES + ((threadIdx.x+blockDim.x*blockIdx.x)% PIVOT_STRIPE_SIZE) ];
      //_learner *shared_learner = &dLearnerMatrix[];
        //_point *feature = &dPointMatrix[0];

        __shared__ _learner shared_learner[16*50];
        int q=threadIdx.x;
             while(q < LEARNER_LENGTH)
             {
             	shared_learner[threadIdx.y*LEARNER_LENGTH+q] = learner[q];
             	q+= blockDim.x;
             }

        //     __syncthreads();
        _learnerBid registers[8];

        registers[0] =0;
        registers[1] =0;
        registers[2] =0;
        registers[3] =0;
        registers[4] =0;
        registers[5] =0;
        registers[6] =0;
        registers[7] =0;

        if(shared_learner[0] < 1){
     //   dLearnerBidMatrix[ learnerId*TOTAL_POINTS+ pointId] = -1 ;//learnerId+LEARNER_LENGTH ;
        }
        else{
        //short progsize = shared_learner[0];
        	int offset = threadIdx.y*LEARNER_LENGTH;
        for (int i=0;i<=shared_learner[0+offset];i++)
        {
            _learnerBid* dst = &registers[((shared_learner[i+offset] & DST_MASK) >> DST_SHIFT)];

            _learnerBid srcVal;

            if (1 == ((shared_learner[i+offset] & MODE_MASK) >> MODE_SHIFT ) %2) {

               // srcVal =  dPointMatrix[threadIdx.x* NUM_FEATURES + ((shared_learner[i] & SRC_MASK) >> SRC_SHIFT) % NUM_FEATURES ];
                srcVal =  feature[(((shared_learner[i+offset] & SRC_MASK) >> SRC_SHIFT) % NUM_FEATURES)*PIVOT_STRIPE_SIZE ];
            }else{
                srcVal =     registers[(((shared_learner[i+offset] & SRC_MASK) >> SRC_SHIFT) % REG_COUNT)];
            }
            switch ( ((shared_learner[i+offset] & OP_MASK) >> OP_SHIFT) % OP_CODE_COUNT){
            case OP_SUM:
                (*dst) += srcVal;
                break;
            case OP_DIFF:
                (*dst) -= srcVal;
                break;
            case OP_PROD:
                (*dst) *= srcVal;
                break;
            case OP_DIV:
                (*dst) /= srcVal;
                break;
            case OP_MOD:
                (*dst) =  fmod((*dst), srcVal);
                break;
            case OP_COS:
                (*dst) = cos(srcVal);
                break;
            case OP_EXP:
                (*dst) = expf(srcVal);
                break;
            case OP_LOG:
                (*dst) = logf(fabs(srcVal));
                break;
            }
            if(isfinite((*dst)) == 0)
                (*dst) = 0;


        }

       	dLearnerBidMatrix[ learnerId*TOTAL_POINTS + pointId] =   1 / (1+exp(-registers[0]));               //<<----------------------
       // dLearnerBidMatrix[ learnerId*TOTAL_POINTS + pointId] = (threadIdx.y < NUM_FEATURES) ? feature[threadIdx.y ] : -1;
		}
    }
}


__host__
void EvaluateLearners(_learner* hLearnerMatrix, _learnerBid* dLearnerBidMatrix,_point* dPointMatrix, int learnerCount, int pointCount)
{


	int xThreads = 16;
	int yThreads = 16;

	int xBlocks = 16;
	int yBlocks = 16;


	_learner* dLearnerMatrix;
    cutilSafeCall( hipMalloc( (void**) &dLearnerMatrix, bytesize_learner * learnerCount   ));
	cutilSafeCall(hipMemcpy (dLearnerMatrix, dLearnerMatrix, bytesize_learner * learnerCount  , hipMemcpyHostToDevice));

	kLearnerEval<<<dim3(xBlocks,yBlocks),dim3(xThreads,yThreads)>>>(0,0,dLearnerMatrix,dLearnerBidMatrix,dPointMatrix,learnerCount, pointCount);



	hipFree(dLearnerMatrix);
}



__host__ void
cLearnerEvalSingle(_learner* learner, _learnerBid &learnerBid, _point* feature, int LEARNER_LENGTH, int NUM_FEATURES )
{

    // printf("cLEArn SINGLE\n");


    float testVal =-1;

    _learnerBid registers[REG_COUNT] ={0};

//		printf("FEATURE ");
//	for(int _a=0;_a< NUM_FEATURES;_a++)
//	{
//		printf(" %f",feature[_a]);
//	}
//		printf("\n");

    short progsize = learner[0];
//     printf(" PROGSIZE: %d" , progsize);
    short val = 0;
    for (int i=1;i<progsize+1;i++)
    {
        unsigned short instruction = learner[i] ;//& 8191;
              // printf( " INST:%d ", instruction);

//		unsigned short inst = instruction;
//		char str[14];
//		str[13] = '\0';
//		for(int p =0 ; p < 13; p++)
//		{
//			str[12-p] = (inst%2==0) ? '0':'1';
//			inst /=2;
//		}
//		printf("%s",str);


        unsigned char mode =0;
        mode = ((instruction & MODE_MASK) >> MODE_SHIFT ) %2;
        char op = 0;
        op = ((instruction & OP_MASK) >> OP_SHIFT) % OP_CODE_COUNT;
        short dst_index = (instruction & DST_MASK) >> DST_SHIFT ; //% REG_COUNT ;
//          printf(" R[%d] <- R[%d] ", dst_index, dst_index);
        _learnerBid* dst = &registers[(instruction & DST_MASK) >> DST_SHIFT];
        unsigned short src = (instruction & SRC_MASK) >> SRC_SHIFT;
        //  float srcVal =	(mode == 1) ? feature[src%NUM_FEATURES] : registers[src%REG_COUNT]   ;
        _learnerBid srcVal;
        // printf(" Mode:%d R[%d]",mode, dst_index );


        if ( mode ==1) {
            srcVal = feature[src%NUM_FEATURES];
//               printf("I[%d] ", src%NUM_FEATURES);
        }else{
            srcVal =     registers[src%REG_COUNT];
//              printf("R[%d] ", src%REG_COUNT);
        }

        //printf(" OP:%d (", op);
        switch (op){
        case OP_SUM:
            (*dst) += srcVal;
//             printf("sum ");
            testVal=srcVal;
            break;
        case OP_DIFF:
            (*dst) -= srcVal;
//                 printf("diff ");
            testVal=1;
            break;
        case OP_PROD:
            (*dst) *= srcVal;
//               printf("prod ");
            testVal=2;
            break;
        case OP_DIV:
            (*dst) /= srcVal;
//               printf("div ");
            testVal=3;
            break;
        case OP_MOD:
            (*dst) =  fmod((*dst), srcVal);
//              printf("mod ");
            testVal=4;
            break;
        case OP_COS:
            (*dst) = cos(srcVal);
//                    printf("cos ");
            break;
        case OP_EXP:
        	//if(srcVal > 88 && srcVal < 709.783){(*dst)=FLT_MAX;}										// Floats Wrapping with EXP -- easiest way to stop it
        	(*dst) = exp(srcVal);
//                   printf("exp ");

            break;
        case OP_LOG:
            (*dst) = logf(fabs(srcVal));
//                printf("log ");
            break;
        }
        if(isfinite((*dst)) == 0)
        {
//        	if( isinf((*dst)) )
//        	{
//        		(*dst) = FLT_MAX;
//        	}
//        	//printf(" DSTNOTFINITE: %f", (*dst) );
//        	else{
            (*dst) = 0;
 //       	}
        }
//                printf("REG ");
//                for ( int i = 0; i < REG_COUNT; i++)
//                {
//
//                    printf("%f ", registers[i]);
//                }
//                printf("\n");
    }
    //        printf(" final:: ");
    //      for ( int i = 0; i < REG_COUNT; i++)
    //      {
    //          printf("   %f ", registers[i]);
    //      }

//    printf(":%f", 1 / float(1+exp(-registers[0])));
    learnerBid = 1 / float(1+exp(-registers[0]));
}





