#include "hip/hip_runtime.h"
#define _DEVICE_VARS_TEST_
#include "GpuTestController.cuh"



#define WARP_SIZE 32
#define MODE_MASK 0x1
#define OP_MASK 0xE
#define DST_MASK 0x70
#define SRC_MASK 0x1F80

#define MODE_SHIFT 0
#define OP_SHIFT 1
#define DST_SHIFT 4
#define SRC_SHIFT 7

#define REG_COUNT 8

#define OP_SUM 0
#define OP_DIFF 1
#define OP_PROD 2
#define OP_DIV 3
#define OP_MOD 4
#define OP_COS 5
#define OP_EXP 6
#define OP_LOG 7
#define OP_CODE_COUNT 8


#define OpCodeCount 7
#define OP_NO 199


#define MAXTHREADS 256


__global__ void kLearnerTest(int val,
                      int learnOffset,
                      int pointOffset,
                      _learner* dLearnerMatrix,
                      _learnerBid* dLearnerBidMatrix,
                      _point* dPointMatrix,
                      int learnerCount,
                      int pointCount)
{
    int pointId = threadIdx.x + blockIdx.x * blockDim.x + pointOffset; // + (blockIdx.x*gridDim.x);
    int learnerId = threadIdx.y + blockIdx.y* blockDim.y + learnOffset;// + blockIdx.y * blockDim.y;
    if (learnerId < TOTAL_LEARNERS && pointId < TOTAL_POINTS)
    {
        int id = threadIdx.x * blockDim.y + threadIdx.y;
        //_learner *shared_learner = &dLearnerMatrix[(threadIdx.y + blockIdx.y*blockDim.y) * LEARNER_LENGTH];
        //_point *feature = &dPointMatrix[(threadIdx.x + blockIdx.x *blockDim.x)*NUM_FEATURES];

        _learner *shared_learner = &dLearnerMatrix[learnerId*LEARNER_LENGTH];
        _point *feature = &dPointMatrix[pointId*NUM_FEATURES];

        //_learner *shared_learner = &dLearnerMatrix[];
        //_point *feature = &dPointMatrix[0];

        __shared__ _learnerBid registers[MAXTHREADS][8];

        registers[id][0] =0;
        registers[id][1] =0;
        registers[id][2] =0;
        registers[id][3] =0;
        registers[id][4] =0;
        registers[id][5] =0;
        registers[id][6] =0;
        registers[id][7] =0;

        //short progsize = shared_learner[0];
        for (int i=1;i<=shared_learner[0];i++)
        {
            _learnerBid* dst = &registers[id][((shared_learner[i] & DST_MASK) >> DST_SHIFT)];

            _learnerBid srcVal;

            if (1 == ((shared_learner[i] & MODE_MASK) >> MODE_SHIFT ) %2) {

               // srcVal =  dPointMatrix[threadIdx.x* NUM_FEATURES + ((shared_learner[i] & SRC_MASK) >> SRC_SHIFT) % NUM_FEATURES ];
                srcVal =  feature[((shared_learner[i] & SRC_MASK) >> SRC_SHIFT) % NUM_FEATURES ];
            }else{
                srcVal =     registers[id][(((shared_learner[i] & SRC_MASK) >> SRC_SHIFT) % REG_COUNT)];
            }
            switch ( ((shared_learner[i] & OP_MASK) >> OP_SHIFT) % OP_CODE_COUNT){
            case OP_SUM:
                (*dst) += srcVal;
                break;
            case OP_DIFF:
                (*dst) -= srcVal;
                break;
            case OP_PROD:
                (*dst) *= srcVal;
                break;
            case OP_DIV:
                (*dst) /= srcVal;
                break;
            case OP_MOD:
                (*dst) =  fmod((*dst), srcVal);
                break;
            case OP_COS:
                (*dst) = cos(srcVal);
                break;
            case OP_EXP:
                (*dst) = expf(srcVal);
                break;
            case OP_LOG:
                (*dst) = logf(fabs(srcVal));
                break;
            }
            if(isfinite((*dst)) == 0)
                (*dst) = 0;


        }

        dLearnerBidMatrix[ threadIdx.y * blockDim.x +  blockIdx.x*blockDim.x + threadIdx.x] =  1 / (1+exp(-registers[id][0]));
         // dLearnerBidMatrix[ threadIdx.y * blockDim.x +  blockIdx.x*blockDim.x + threadIdx.x] = val+1;
        }

}

__global__ void kEval2(int val,
                      int learnOffset,
                      int pointOffset,
                      _learner* dLearnerMatrix,
                      _learnerBid* dLearnerBidMatrix,
                      _point* dPointMatrix,
                      int learnerCount,
                      int pointCount)
{
    int pointId = threadIdx.x + blockIdx.x * blockDim.x + pointOffset ;
    int learnerId = threadIdx.y + blockIdx.y* blockDim.y + learnOffset ;
   // int learnerId = threadIdx.y  + blockIdx.y * blockDim.y ;
    if (learnerId < TOTAL_LEARNERS && pointId < TOTAL_POINTS)
    {
      //   dLearnerBidMatrix[ learnerId*TOTAL_POINTS + pointId] = dPointMatrix[pointId * NUM_FEATURES + learnerId] ; // + min(LEARNER_LENGTH-1, pointId)];// blockIdx.x * gridDim.y + blockIdx.y + 1;

      //  dLearnerBidMatrix[ learnerId*TOTAL_POINTS + pointId] = 1 ;//learnOffset;//blockIdx.x * gridDim.y + blockIdx.y + 1;
        dLearnerBidMatrix[ threadIdx.y * blockDim.x +  threadIdx.x] = threadIdx.x;//blockIdx.x * gridDim.y + blockIdx.y + 1;

    }
}

__host__
void TestLearners(	int learnerCount,
					int learnerLength,
					_learner* hLearnerMatrix,
					_learnerBid* hLearnerBidMatrix,
					int trainingSetSize,
					_point* hTrainingSet ,
					int numFeatures
				 )
{

	int bytesize_singlePoint = sizeof(_point) * numFeatures;
	int bytesize_learner = sizeof(_learner) * learnerLength;
    //////////////////////////
	// Memory Allocations
	//////////////////////////
	int streamCount = 2;


	_point* dTrainingSet;
	_learner* dLearnerMatrix;
	_learnerBid* dBidMatrix[streamCount];

	 int learnersPerChunk =1 ;
	 int pointsPerChunk = 512;


	 cutilSafeCall( hipMalloc( (void**) &dTrainingSet, bytesize_singlePoint * trainingSetSize ));
	 cutilSafeCall( hipMalloc( (void**) &dLearnerMatrix, bytesize_learner * learnerCount ));

	 hipStream_t* stream = new hipStream_t[streamCount ];
	 	for(int i=0; i < streamCount; i++)
	 	{
	 		cutilSafeCall( hipMalloc( (void**) &dBidMatrix[i], sizeof(_learnerBid) * learnersPerChunk* pointsPerChunk ));
	 		hipStreamCreate(&stream[i]);
	 	}

	// cutilSafeCall( hipMalloc( (void**) &dBidMatrix[1], sizeof(_learnerBid) * learnersPerChunk* pointsPerChunk ));

	 hipMemset(dLearnerMatrix,0,bytesize_learner * learnerCount );
	 hipMemset(dBidMatrix[0],0,sizeof(_learnerBid) * learnersPerChunk * pointsPerChunk  );

    cutilSafeCall( hipMemcpyToSymbol(HIP_SYMBOL( TOTAL_LEARNERS), &learnerCount,sizeof(short)));
    cutilSafeCall( hipMemcpyToSymbol(HIP_SYMBOL( TOTAL_POINTS), &trainingSetSize,sizeof(short)));

    cutilSafeCall( hipMemcpyToSymbol(HIP_SYMBOL( NUM_FEATURES), &numFeatures,sizeof(short)));
    cutilSafeCall( hipMemcpyToSymbol(HIP_SYMBOL( LEARNER_LENGTH), &learnerLength,sizeof(short)));


     int learnerChunkCount = (learnerCount-1)/ learnersPerChunk+1;
     int pointChunkCount = (trainingSetSize-1) / pointsPerChunk+1;

    int pointsPerBlock = 256;
     int learnersPerBlock = min(MAXTHREADS/pointsPerBlock , learnerCount);
     int threadsPerBlock_x = pointsPerBlock ;
     int threadsPerBlock_y = learnersPerBlock;



     int blocksPerGrid_x = ((pointsPerChunk-1)/pointsPerBlock+1);
     int blocksPerGrid_y = ((learnersPerChunk-1)/learnersPerBlock+1);

     dim3 GRID (blocksPerGrid_x,blocksPerGrid_y);
     dim3 BLOCK (threadsPerBlock_x,threadsPerBlock_y);



  //   printf(" ###  lpC:%d   lCC:%d     ppC:%d     pCC:%d  PS:%d   BPGx%d   BPGy:%d\n" , learnersPerChunk , learnerChunkCount, pointsPerChunk, pointChunkCount , trainingSetSize, blocksPerGrid_x ,blocksPerGrid_y );

    cutilSafeCall(hipMemcpy (dLearnerMatrix, hLearnerMatrix, learnerCount* bytesize_learner, hipMemcpyHostToDevice));
    cutilSafeCall(hipMemcpy (dTrainingSet, hTrainingSet, trainingSetSize* bytesize_singlePoint, hipMemcpyHostToDevice));
//

    int streamA = 0;
    int streamB = 1;

    for(int chunkId_y=0; chunkId_y< learnerChunkCount;chunkId_y ++)
    {

     	 for(int chunkId_x=0; chunkId_x< pointChunkCount;chunkId_x +=streamCount)
          {
     		 //kLearnerTest<<<GRID,BLOCK,0,stream[streamId]>>>(chunkId_x,chunkId_y        ,chunkId_x*pointsPerChunk,dLearnerMatrix,dBidMatrix[0], dTrainingSet, 1, trainingSetSize);
     		 for(int streamId=0; streamId < streamCount; streamId++)
     		 {
     			 if(chunkId_x+streamId < pointChunkCount){
     				 kLearnerTest<<<GRID,BLOCK,0,stream[streamId]>>>(chunkId_x+streamId,chunkId_y        ,(chunkId_x+streamId)*pointsPerChunk,dLearnerMatrix,dBidMatrix[streamId], dTrainingSet, 1, trainingSetSize);
     			 }
     		 }

     		for(int streamId=0; streamId < streamCount; streamId++)
     		     		 {
     		     			 if(chunkId_x+streamId < pointChunkCount){
     		     				cutilSafeCall(hipMemcpyAsync (hLearnerBidMatrix + chunkId_y*trainingSetSize + (chunkId_x+streamId)* pointsPerChunk , dBidMatrix[streamId], pointsPerChunk * sizeof(_learnerBid), hipMemcpyDeviceToHost,stream[streamId]));
     		     			 }
     		     		 }

          }
     	// int offset = chunkId_y * learnersPerChunk * trainingSetSize;
  		//cutilSafeCall(hipMemcpyAsync (hLearnerBidMatrix + chunkId_y*trainingSetSize , dBidMatrix[0], trainingSetSize * sizeof(_learnerBid), hipMemcpyDeviceToHost,stream[streamB]));


      }

	hipDeviceSynchronize();

    cutilSafeCall( hipFree( dLearnerMatrix));
    cutilSafeCall( hipFree( dBidMatrix[0]));
    //cutilSafeCall( hipFree( dBidMatrix[1]));
    cutilSafeCall( hipFree( dTrainingSet));

    for(int i=0; i < streamCount; i++)
    	{
    		hipStreamDestroy(stream[i]);
    	}

}
