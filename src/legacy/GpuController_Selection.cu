#include "hip/hip_runtime.h"
#define _DEVICE_VARS_

#define DEBUGMODE
#include "GpuController_Selection.cuh"

  #include <thrust/logical.h>
  #include <thrust/functional.h>

long Diff2(timeval tv_start, timeval tv_end){
   return 1000000*(tv_end.tv_sec - tv_start.tv_sec) + tv_end.tv_usec - tv_start.tv_usec;
}

#define BIGNUMBER 99999;
#define STATE_FRONT_TOO_SMALL 0
#define STATE_FRONT_TOO_BIG 1
#define STATE_FRONT_IS_JUUUST_RIGHT 2

int MSB(int v)
{
const unsigned int b[] = {0x2, 0xC, 0xF0, 0xFF00, 0xFFFF0000};
const unsigned int S[] = {1, 2, 4, 8, 16};
int i;

register unsigned int r = 0; // result of log2(v) will go here
for (i = 4; i >= 0; i--) // unroll for speed...
{
  if (v & b[i])
  {
    v >>= S[i];
    r |= S[i];
  }
}
return r;
}

int nextPowerOf2(int v)
{
	return 1<<(MSB(v)+1);
}


__global__ void kCalcDist(_teamReward* rewards, _teamReward* dist_out, int teamCount, int pointCount)
{
	if(threadIdx.x < teamCount && blockIdx.x < pointCount)
	{
		_teamReward* dist;
	//	_teamReward* rewards;
		int j = threadIdx.x;

		dist = dist_out +  blockIdx.x*teamCount*teamCount;

		for(int i=0;i < teamCount; i++)
		{
				dist[j+i*pointCount] = (rewards[i*teamCount+blockIdx.x] > rewards[j*teamCount+blockIdx.x]);

			//	if(j==1){dist[j+i*teamCount] = 1;}
		}

	}
}
// lets assume 16x16 for fun
__global__ void kCalcDistSoMuchBetterBro(_teamReward* rewards, _teamReward* dist_out, int teamCount, int pointCount)
{
	int pointIndex = blockIdx.z;
	int i = blockIdx.y*blockDim.y + threadIdx.y;
	int j = blockIdx.x*blockDim.x + threadIdx.x;

//	extern __shared__ _teamReward XCache[];					GAH You 2 hours debug later: TL;DR can't have two separate arrays
//	extern __shared__ _teamReward YCache[];
	extern __shared__ _teamReward Cache[];
	if(i < teamCount && j < teamCount && pointIndex < pointCount)
	{

			if(0 == threadIdx.y )
			{
				Cache[threadIdx.x] = rewards[j*pointCount+pointIndex];
			}
			if(0 == threadIdx.x )
			{
				Cache[threadIdx.y+blockDim.x] = rewards[i*pointCount+pointIndex];
			}

	}
		__syncthreads();
		if(i < teamCount && j < teamCount && pointIndex < pointCount)
		{
			dist_out[pointIndex*teamCount*teamCount + i*teamCount+j] =  Cache[threadIdx.y+blockDim.x] > Cache[threadIdx.x];
		}





}


__host__ void GetDist(_teamReward* rewards, _teamReward* dist_out, int teamCount, int pointCount)
{
	int threads = 8;
	int xThreads = threads;
	int yThreads = threads;

	int xBlocks = (teamCount-1) / xThreads +1;
	int yBlocks = (teamCount-1) / yThreads +1;
	int zBlocks = pointCount;


	hipMemset(dist_out,0,sizeof(_teamReward) * teamCount*teamCount*pointCount);

	if(xThreads*yThreads > 1024){ fprintf(stderr,"Error: Too many threads used in GetDist");}
	kCalcDistSoMuchBetterBro<<<dim3(xBlocks,yBlocks,zBlocks),dim3(xThreads,yThreads),xThreads+yThreads>>>(rewards, dist_out, teamCount, pointCount);
	cutilCheckMsg("Kernel execution failed");//?? WTF???A



}


__global__ void isDominated___(_teamReward* Vec, int size, int vecCount,bool* DOM, bool* EQUAL, int* OUT)
{

	int isIndex = blockIdx.x;
	int byIndex = blockIdx.y;


		_teamReward* vIs= &Vec[size*isIndex];
		_teamReward* vBy= &Vec[size*byIndex];
		__shared__ bool dominated[128];
		__shared__ bool equal[128];

		__shared__ bool isDominated;
		__shared__ bool isEqual;
		if(threadIdx.x ==0){isDominated = true; isEqual =true;}
		__syncthreads();

		int elementOffset = 0;
		while(isDominated && elementOffset*blockDim.x < size)
		{
			int tid = 0;
			tid = threadIdx.x + elementOffset;


			//int active_threads = (size - stepIndex*blockDim.x > blockDim.x) ? blockDim.x : size ;
			if(tid < size)
			{
				dominated[threadIdx.x] = (vIs[tid] > vBy[tid]);
				equal[threadIdx.x] = (vIs[tid] == vBy[tid]);
			}else{
				dominated[threadIdx.x] = 0;
				equal[threadIdx.x] = 1;
			}

			__syncthreads();

			int activeThreads = blockDim.x/2;

			while(activeThreads >0)
			{
				if(threadIdx.x < activeThreads && tid+activeThreads < size)
				{
					dominated[threadIdx.x] = dominated[threadIdx.x] || dominated[threadIdx.x+activeThreads];
					equal[threadIdx.x] = equal[threadIdx.x] && equal[threadIdx.x+activeThreads];
				}

				__syncthreads();
				activeThreads /=2;
			}

		__syncthreads();
			if(threadIdx.x ==0)
			{
				isDominated = !dominated[0];
				isEqual = isEqual && equal[0];
					//DOM[isIndex*blockDim.y + byIndex] = dominates[0];
					//EQUAL[isIndex*blockDim.y + byIndex] = equal[0];
			}

			elementOffset += blockDim.x;



		}

		__syncthreads();

		if(threadIdx.x ==0)
		{

			//EQUAL[isIndex*vecCount + byIndex] = isEqual;
			DOM[isIndex*vecCount + byIndex] = (byIndex < isIndex && isEqual) || (isDominated && !isEqual);
			OUT[isIndex*vecCount + byIndex] =  dominated[0];
		}



}
__global__ void isDominated(_teamReward* Vec, int size, int vecCount,bool* DOM, bool* EQUAL, int* OUT)
{

	int isIndex = blockIdx.x;
	int byIndex = blockIdx.y;


		_teamReward* vIs= &Vec[size*isIndex];
		_teamReward* vBy= &Vec[size*byIndex];
		__shared__ bool dominated[128];
		__shared__ bool equal[128];

		__shared__ bool isDominated;
		__shared__ bool isEqual;
		if(threadIdx.x ==0){isDominated = true; isEqual =true;}
		__syncthreads();

		int elementOffset = 0;
		while(isDominated && elementOffset*blockDim.x < size)
		{
			int tid = 0;
			tid = threadIdx.x + elementOffset;


			//int active_threads = (size - stepIndex*blockDim.x > blockDim.x) ? blockDim.x : size ;
			if(tid < size)
			{
				dominated[threadIdx.x] = (vIs[tid] > vBy[tid]);
				equal[threadIdx.x] = (vIs[tid] == vBy[tid]);
			}else{
				dominated[threadIdx.x] = 0;
				equal[threadIdx.x] = 1;
			}

			__syncthreads();

			int activeThreads = blockDim.x/2;

			while(activeThreads >0)
			{
				if(threadIdx.x < activeThreads && tid+activeThreads < size)
				{
					dominated[threadIdx.x] = dominated[threadIdx.x] || dominated[threadIdx.x+activeThreads];
					equal[threadIdx.x] = equal[threadIdx.x] && equal[threadIdx.x+activeThreads];
				}

				__syncthreads();
				activeThreads /=2;
			}

		__syncthreads();
			if(threadIdx.x ==0)
			{
				isDominated = !dominated[0];
				isEqual = isEqual && equal[0];
					//DOM[isIndex*blockDim.y + byIndex] = dominates[0];
					//EQUAL[isIndex*blockDim.y + byIndex] = equal[0];
			}

			elementOffset += blockDim.x;



		}

		__syncthreads();

		if(threadIdx.x ==0)
		{

			//EQUAL[isIndex*vecCount + byIndex] = isEqual;
			DOM[isIndex*vecCount + byIndex] = (byIndex < isIndex && isEqual) || (isDominated && !isEqual);
			OUT[isIndex*vecCount + byIndex] =  isDominated;
		}



}


// TODO: BRO fix  this -- Gonn ahve to loop over an do a recursive reduction
__global__ void findParetoFront(bool* DOM, bool* front, int size)
{
	int pointId = blockIdx.y + blockIdx.x*gridDim.x;
	int index = threadIdx.x;


	extern __shared__ bool data[];



	int offset = blockDim.x;
	data[threadIdx.x] = ((index < size) ? DOM[pointId*size+index]: 0) or ((index + offset < size) ? DOM[pointId*size + index + offset] : 0);



	// Reduce
	__syncthreads();
	int activeThreads = blockDim.x/2;
	while(activeThreads > 0)
	{
		if(threadIdx.x < activeThreads)
			data[threadIdx.x] = data[threadIdx.x] or data[threadIdx.x + activeThreads];
		__syncthreads();
		activeThreads /= 2;
	}

	__syncthreads();
	if(0 == threadIdx.x )
		front[pointId] = data[0];


}

__global__ void findParetoFront2(bool* DOM, bool* out,int size, int pointOffset, int* out2)
{
	int rowId = blockIdx.y ;//+ pointOffset;
	int index = threadIdx.x ;//+ blockIdx.x*blockDim.x;


	extern __shared__ bool data[];


	int offset = blockDim.x;
	data[threadIdx.x] = ((index < size) ? DOM[rowId*size+index]: 0) or ((index + offset < size) ? DOM[rowId*size + index + offset] : 0);


	// Reduce
	__syncthreads();
	int activeThreads = blockDim.x/2;
	while(activeThreads > 0)
	{
		if(threadIdx.x < activeThreads)
			data[threadIdx.x] = data[threadIdx.x] or data[threadIdx.x + activeThreads];
		__syncthreads();
		activeThreads /= 2;
	}

	__syncthreads();
	if(0 == threadIdx.x )
	{
		out[rowId*gridDim.x+blockIdx.x] = !data[0];
	}



}
__global__ void col2Row(bool* to, bool* from,int rowCount,int colCount)
{
	int a;
	if( threadIdx.x < rowCount)
	{
		to[threadIdx.x] = from[threadIdx.x*colCount];
	}

}

__host__ void FindParetoFront(bool* dom, bool* front,int size)	// PS its  rowSize * rowSize Matrix
{
	int rowsPerInvocation = size;

	int colCount = size;
	int rowCount = size;

	int xThreads = 128;
	int xBlocks = (colCount-1) / xThreads +1;
	int yBlocks = rowCount;


	cutilCheckMsg("Pre Kernel execution failed");//?? WTF???A

//	bool* A = new bool[rowCount*colCount];
//	cutilSafeCall(hipMemcpy (A, dom, sizeof(bool) * colCount * rowCount, hipMemcpyDeviceToHost));
//
//	printf("====^^^^^====\n");
//		for(int i =0; i < rowCount; i++)
//		{
//			for(int j=0;j<colCount;j++)
//			{
//				printf(" %s" , (A[i*colCount+j]) ? "1":"0" );
//			}
//			printf("\n");
//		}

		int colsLeft = colCount;



	bool* dStaging;
	cutilSafeCall( hipMalloc( (void**) &dStaging,sizeof(bool) * rowCount * xBlocks   ));
	int* tmp;
		cutilSafeCall( hipMalloc( (void**) &tmp,sizeof(int) * rowCount * rowCount   ));
	//	cutilSafeCall(hipMemcpy (dWorking, dData, sizeof(int) * colCount *rowCount, hipMemcpyDeviceToDevice));

	int rowOffset = 0;

						// TODO: Which way? X blocks First or Y Block First (X = finnish Summation? Y =Finish all rows first
	while(colsLeft > 1)
	{
		xBlocks = (colsLeft-1)/xThreads+1;

		while(rowOffset < yBlocks)
		{

	//		printf("BSS %d\n", xBlocks);
			int blocksPerGrid_x =  xBlocks;
			int blocksPerGrid_y = (rowCount > rowsPerInvocation) ? rowsPerInvocation : rowCount;

			findParetoFront2<<<dim3(1,blocksPerGrid_y),xThreads,xThreads>>>(dom,dStaging,colsLeft,rowOffset,tmp);
			cutilCheckMsg("Kernel execution failed");//?? WTF???A


			rowOffset += blocksPerGrid_y;
		}
		colsLeft = xBlocks;
	}
		int t[rowCount*rowCount];
//		cutilSafeCall(hipMemcpy (t, tmp, sizeof(int) * rowCount *rowCount, hipMemcpyDeviceToHost));
//		cutilSafeCall(hipMemcpy (A, dStaging, sizeof(bool) * xBlocks *rowCount, hipMemcpyDeviceToHost));
//		printf("====TMP====");
//		for(int i =0; i < rowCount; i++)
//				{
//					for(int j=0;j< rowCount;j++)
//					{
//						printf(" %d" , t[i*rowCount+j]);
//					}
//					printf("\n");
//				}
//		printf("====#====");
//		for(int i =0; i < rowCount; i++)
//		{
//			for(int j=0;j< xBlocks;j++)
//			{
//				printf(" %s" , (A[i*xBlocks+j]) ? "1":"0" );
//			}
//			printf("\n");
//		}
//
//	printf(" R == %d\n" ,nextPowerOf2(1204));
	col2Row<<<1,nextPowerOf2(rowCount)>>>(front,dStaging,rowCount,xBlocks);					// LIMIT: rowCount == 1024
//	cutilSafeCall(hipMemcpy (front, dStaging, sizeof(bool) *rowCount, hipMemcpyDeviceToDevice));

//	cutilSafeCall(hipMemcpy (A, front, sizeof(bool) * rowCount, hipMemcpyDeviceToHost));
//		printf("========");
//		for(int i =0; i < rowCount; i++)
//		{
////			for(int j=0;j< xBlocks;j++)
////			{
//				printf(" %s" , (A[i]) ? "1":"0" );
////			}
//			printf("\n");
//		}
//
//

//	delete A;


}
__global__ void kColumnSum(int* vec, int vecCount, int rowCount,int* out, int offset)
{
	int vecIndex = blockIdx.x + offset;
	int rowIndex = threadIdx.y + blockIdx.y*blockDim.y;
	__shared__ int sum_vec[256];

	sum_vec[threadIdx.y] = 0;

	if(rowIndex < rowCount)
	{
		sum_vec[threadIdx.y] = vec[vecIndex+ rowIndex*vecCount]; //vec[vecIndex + pointCount*vecCount ];
	}



int i = blockDim.y;
i /=2 ;
while(i>0)
{
	__syncthreads();
	if( threadIdx.y < i  )
	{
		sum_vec[threadIdx.y] = sum_vec[threadIdx.y]+ sum_vec[threadIdx.y+ i];//threadIdx.y* 1000 + threadIdx.y+(i/2);

	}
	i /=2 ;

}


	__syncthreads();

	if(threadIdx.y==0)
	{
		out[vecIndex + blockIdx.y*vecCount] = sum_vec[0];

	}


}

__host__ void ColumnSum(int* vec, int rowCount, int colCount)
{
	printf("ColSum - R:%d C:%d ", rowCount, colCount);
	int yThreads = 128;
	int yBlocks = (rowCount -1)/ yThreads +1;
	cutilCheckMsg("Pre Kernel execution failed");//?? WTF???A
	int blocksPerGrid_x = (colCount > 40) ? 40 : colCount;
	for(int _rowsLeft=rowCount; _rowsLeft >1;_rowsLeft = (_rowsLeft -1) / yThreads+1)
	{
		printf("rowCount: %d   %d %d\n",rowCount, colCount,yBlocks);
		int elementsRemaining = colCount;
		for(int offset = 0; offset+blocksPerGrid_x <= colCount; offset += blocksPerGrid_x )
		{
			printf("asdasDasd %d\n", offset);
			kColumnSum <<<dim3(blocksPerGrid_x,yBlocks),dim3(1,yThreads)>>>(vec,colCount, rowCount, vec,offset);
			cutilCheckMsg("Kernel execution failed");//?? WTF???A
		}

	}
}

template< typename T >
__global__ void kRowSum(T* vec, int rowCount, int vecCount,T* out, int offset)
{
	int vecIndex = threadIdx.x + blockIdx.x*blockDim.x;
	int rowIndex = blockIdx.y + offset;
	__shared__ T sum_vec[256];

	sum_vec[threadIdx.x] = 0;

//	if(rowIndex < rowCount)
//	{
//		sum_vec[threadIdx.x] = vec[vecIndex+ rowIndex*vecCount] ; //vec[vecIndex + pointCount*vecCount ];
//	}
	sum_vec[threadIdx.x] = ((vecIndex < vecCount) ?  vec[vecIndex+ rowIndex*vecCount]: 0); //or ((index + offset < size) ? DOM[pointId*size + index + offset] : 0);


	int i = blockDim.x/2;
	while(i > 0)
	{
		__syncthreads();
		if(threadIdx.x <i)
			sum_vec[threadIdx.x] = sum_vec[threadIdx.x]+ sum_vec[threadIdx.x+ i];//threadIdx.y* 1000 + threadIdx.y+(i/2);
		i /=2;
	}



	__syncthreads();

	if(threadIdx.x  < vecCount)
	{
		out[vecIndex + rowIndex*vecCount] = sum_vec[threadIdx.x];

	}


}
__host__ void RowSum(float* dVec, int rowCount, int colCount)
{
	printf("RowSum - R:%d C:%d ", rowCount, colCount);
	int xThreads = 128;
	int xBlocks =  (colCount -1)/ xThreads +1;
	int blocksPerGrid_y = (rowCount > 40) ? 40 : rowCount;
	for(int _rowsLeft=colCount; _rowsLeft >1;_rowsLeft = (_rowsLeft -1) / xThreads+1)
	{
		printf(" XB:%d XT:%d\n",xBlocks, xThreads);
		int elementsRemaining = colCount;
		for(int offset = 0; offset+blocksPerGrid_y <= rowCount; offset += blocksPerGrid_y )
		{
			printf("asdasDasd\n");
			kRowSum <<<dim3(xBlocks,blocksPerGrid_y),dim3(xThreads,1)>>>(dVec,rowCount, colCount, dVec,offset);

			cutilCheckMsg("Kernel execution failed");//?? WTF???A
		}

	}
}
template< typename T >
__global__ void FrontFilter(T* vec, bool* front, bool Bool, T val, int vecCount, int pointCount, T* out,int offset)
{
	int vecIndex = blockIdx.x + offset;
	int pointIndex = threadIdx.y + blockIdx.y*blockDim.y;

	if(pointIndex < pointCount )
	{
		if(Bool != front[pointIndex])
		{
			out[vecIndex + pointIndex * vecCount] = val;
		}
		//out[vecIndex + pointIndex * vecCount] = (Bool == front[pointIndex]) ? vec[vecIndex + pointCount*vecCount ] : 0;
	}

}



void PTeamSelectParetoSerial(bool mode, bool* front, float* scores, int rowCount, int colCount,int _frontCount, int nrem, thrust::host_vector<int>* toDel)
{

	  struct timeval tv_1;
		struct timeval tv_2;
		struct timezone tz;
		long timer0 = 0;
		gettimeofday(&tv_1, &tz);
	//printf("SIZE TODEL: %d\n", toDel->size());
	//Sort
	thrust::host_vector<int> hIndex(0);
	//thrust::sequence(hIndex.begin(), hIndex.end());
	thrust::host_vector<float> hScores(0);


	thrust::host_vector<int> hFrontMask(rowCount);
	thrust::device_vector<int> dFrontMask(rowCount);
	int frontCount =0;
	//printf("IDESCAORES: ");
	for(int i=0;i<rowCount;i++)
	{
		if(front[i]== !mode)
		{
			frontCount++;
			hIndex.push_back(i);
			hScores.push_back(scores[i*colCount]);
	//		printf(" %f", scores[i*colCount]);
		//	hFrontMask.push_back(front[i]);
		}

	}
//
	//printf("\nNot Sorted  ");
		for(int i =0; i <hIndex.size();i++)
			{
				printf("  $%d::%f ",hIndex[i], hScores[i]);
			}
		printf("\n");
	for(int i= hIndex.size()-1; i > 0 ;i--)
	{
		for(int j=0; j < i;j++)
		{
			if(hScores[j] > hScores[j+1])
			{
				int a = hIndex[j];
				float b = hScores[j];

				hIndex[j] = hIndex[j+1];
				hScores[j] = hScores[j+1] ;

				hIndex[j+1] = a;
				hScores[j+1] = b ;
			}
		}
	}
printf("\nSoreted  ");
	for(int i =0; i <hIndex.size();i++)
		{
			printf("  $%d::%f ",hIndex[i], hScores[i]);
		}
	printf("\n");

	for(int i=0; i < nrem;i++)
	{
		toDel->push_back(hIndex[i]);
	}

	//dFrontMask = hFrontMask;
	//int result = thrust::count(dFrontMask.begin(), dFrontMask.end(), 1);

	//printf("FrontCount = %d vs %d",count,result );


//	for(int i=0; i < pointCount;i++)
//	{
//		printf("  %d:%f", i,scores[i]);
//	}
//	printf("\n");




	// Find NLowest
	gettimeofday(&tv_2, &tz);
		timer0 = Diff2(tv_1,tv_2);
		printf(" Stage Pareto  : %ld\n", timer0 );
}
__global__ void kElementDivide(int* A, int* B, int vecCount, float* out)
{
	if(threadIdx.x + blockIdx.x*blockDim.x < vecCount)
	out[threadIdx.x + blockIdx.x*blockDim.x] = (A[threadIdx.x +blockIdx.x*blockDim.x]+1) /  (float(B[threadIdx.x + blockIdx.x*blockDim.x])+1);
}

__global__ void kColumnNormalize(int* dData,int* dBaseSum,int rowCount, int colCount, float* dOut)
{
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	extern __shared__ int divisor[];

	if(row< rowCount && col < colCount)
	{
	if(threadIdx.y == 0)
	{
		divisor[threadIdx.x] = dBaseSum[col];
	}
	}
	__syncthreads();
	if(row< rowCount && col < colCount)
		{
	dOut[row*colCount+col] = float(dData[row*colCount+col])/divisor[threadIdx.x];
	if(false == isfinite(dOut[row*colCount+col])){ dOut[row*colCount+col] = 0;}
		}

}


__host__ void frontToVector(bool* front,int rowCount,thrust::host_vector<int>* hToDel  )
{
	int count =0;
	for(int i=0;i<rowCount;i++){
		if(!front[i]){
			(*hToDel)[count++]=i;
		}
	}
}

__host__ void calcScores_D(_teamReward* dData, bool* dFront, float* dScores, int rowCount, int colCount ,int frontCount)
{




	int yThreads = 128;
	int yBlocks = (rowCount -1)/ yThreads +1;

	int* dBaseSum;
	int* dASD;


	cutilSafeCall( hipMalloc( (void**) &dBaseSum,sizeof(int) * colCount * rowCount   ));
	cutilSafeCall(hipMemcpy (dBaseSum, dData, sizeof(int) * colCount *rowCount, hipMemcpyDeviceToDevice));


	int blocksPerGrid_x = (colCount > 40) ? 40 : colCount;

	ColumnSum(dBaseSum,rowCount,colCount);

	// >>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>
//	int data[rowCount*colCount];
//	cutilSafeCall(hipMemcpy (data, dData, sizeof(int) * rowCount * colCount, hipMemcpyDeviceToHost));
//
//	for(int i=0;i < rowCount; i++)
//	{
//		for(int j = 0; j < colCount;j++)
//		{
//			printf(" %d", data[j+i *colCount]);
//		}
//		printf("\n");
//	}
//
//	int distSum[colCount];
//	cutilSafeCall(hipMemcpy (distSum, dBaseSum, sizeof(int) * colCount, hipMemcpyDeviceToHost));
//	printf("DataSUM\n");
//	for(int i = 0; i < colCount; i++)
//	{
//		printf(" %d ", distSum[i]);
//	}
//	printf("\n");
//
//	int* dFrontSum;

	// <<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<
	int* dWorking;
	cutilSafeCall( hipMalloc( (void**) &dWorking,sizeof(int) * colCount * rowCount   ));
	cutilSafeCall(hipMemcpy (dWorking, dData, sizeof(int) * colCount *rowCount, hipMemcpyDeviceToDevice));

	int threads = 16;
	kColumnNormalize<<<dim3((colCount-1)/threads+1,(rowCount-1)/threads+1),dim3(threads,threads),threads>>>(dWorking,dBaseSum,rowCount,colCount,dScores);



	for(int offset = 0; offset+blocksPerGrid_x <= colCount; offset += blocksPerGrid_x )
	{
	//	printf("adsafghgh \n");
		FrontFilter<<<dim3(blocksPerGrid_x,yBlocks),dim3(1,yThreads)>>>(dScores,dFront,0,(float)6.44,colCount, rowCount, dScores, offset);
		cutilCheckMsg("Kernel execution failed");
	}

	// >>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>
//	float frontSum[colCount*rowCount];
//		cutilSafeCall(hipMemcpy (frontSum, dScores, sizeof(float) * colCount * rowCount, hipMemcpyDeviceToHost));
//		printf("FORNTSUM\n");
//		for(int i = 0; i < rowCount; i++)
//		{
//			for(int j = 0; j < colCount; j++)
//					{
//			printf(" %0.2f ", frontSum[j + i*colCount ]);
//		}
//		printf("\n");
//		}

	// <<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<


	RowSum(dScores,rowCount,colCount);
//	for(int _rowsLeft=rowCount; _rowsLeft >1;_rowsLeft = (_rowsLeft -1) / yThreads+1)
//	{
//		printf("rowCount: %d   %d %d\n",rowCount, colCount,yBlocks);
//		int elementsRemaining = colCount;
//		for(int offset = 0; offset+blocksPerGrid_x <= colCount; offset += blocksPerGrid_x )
//		{
//			printf("asdasDasd\n");
//			kColumnSum <<<dim3(blocksPerGrid_x,yBlocks),dim3(1,yThreads)>>>(dFrontSum,colCount, rowCount, dFrontSum,offset);
//			cutilCheckMsg("Kernel execution failed");//?? WTF???A
//		}
//
//	}

	// >>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>
	//int frontSum[colCount];
//	cutilSafeCall(hipMemcpy (frontSum, dScores, sizeof(float) * colCount * rowCount, hipMemcpyDeviceToHost));
//		printf("FORNTSUM\n");
//		for(int j = 0; j < rowCount; j++)
//		{
//			for(int i = 0; i < colCount; i++)
//				printf(" %0.2f ", frontSum[i+j*colCount]);
//			printf("\n");
//		}
//		printf("\n");

	// <<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<

//	int xThreads = 128;
//	//float* dScore;
//	//cutilSafeCall( hipMalloc( (void**) &dScore,sizeof(float) *colCount   ));
//	kElementDivide<<<dim3((colCount-1)/xThreads+1,1),dim3(xThreads)>>>(dFrontSum,dBaseSum,colCount,dScores);
//	cutilCheckMsg("Kernel execution failed");

//
//
//	cutilSafeCall(hipFree( dDistSum));
//	cutilSafeCall(hipFree( dFrontSum));
//	cutilSafeCall(hipFree( dScore));
	hipFree(dWorking);
	hipFree(dBaseSum);
}


__host__ void calcScores_F(_teamReward* dData, bool* dFront, float* dScores, int rowCount, int colCount ,int frontCount)
{
	int state = STATE_FRONT_TOO_BIG;

	int yThreads = 128;
	int yBlocks = (rowCount -1)/ yThreads +1;

	int* dBaseSum;
	int* dASD;


	cutilSafeCall( hipMalloc( (void**) &dBaseSum,sizeof(int) * colCount * rowCount   ));
	cutilSafeCall(hipMemcpy (dBaseSum, dData, sizeof(int) * colCount *rowCount, hipMemcpyDeviceToDevice));


	int blocksPerGrid_x = (colCount > 40) ? 40 : colCount;
	if(state == STATE_FRONT_TOO_BIG)
	{
	for(int offset = 0; offset+blocksPerGrid_x <= colCount; offset += blocksPerGrid_x )
		{
			FrontFilter<<<dim3(blocksPerGrid_x,yBlocks),dim3(1,yThreads)>>>(dBaseSum,dFront,1,0,colCount, rowCount, dBaseSum, offset);
			cutilCheckMsg("Kernel execution failed");
		}
	}
	ColumnSum(dBaseSum,rowCount,colCount);



	// >>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>
//	int data[rowCount*colCount];
//	cutilSafeCall(hipMemcpy (data, dData, sizeof(int) * rowCount * colCount, hipMemcpyDeviceToHost));
//
//	for(int i=0;i < rowCount; i++)
//	{
//		for(int j = 0; j < colCount;j++)
//		{
//			printf(" %d", data[j+i *colCount]);
//		}
//		printf("\n");
//	}
//
//	int distSum[colCount];
//	cutilSafeCall(hipMemcpy (distSum, dBaseSum, sizeof(int) * colCount, hipMemcpyDeviceToHost));
//	printf("DataSUM\n");
//	for(int i = 0; i < colCount; i++)
//	{
//		printf(" %d ", distSum[i]);
//	}
//	printf("\n");
//
//	int* dFrontSum;

	// <<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<
	int* dWorking;
	cutilSafeCall( hipMalloc( (void**) &dWorking,sizeof(int) * colCount * rowCount   ));
	cutilSafeCall(hipMemcpy (dWorking, dData, sizeof(int) * colCount *rowCount, hipMemcpyDeviceToDevice));

	int threads = 16;
	kColumnNormalize<<<dim3((colCount-1)/threads+1,(rowCount-1)/threads+1),dim3(threads,threads),threads>>>(dWorking,dBaseSum,rowCount,colCount,dScores);

	for(int offset = 0; offset+blocksPerGrid_x <= colCount; offset += blocksPerGrid_x )
	{
		FrontFilter<<<dim3(blocksPerGrid_x,yBlocks),dim3(1,yThreads)>>>(dScores,dFront,STATE_FRONT_TOO_BIG==state,(float)6.44,colCount, rowCount, dScores, offset);
		cutilCheckMsg("Kernel execution failed");
	}

	// >>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>
//	float frontSum[colCount*rowCount];
//		cutilSafeCall(hipMemcpy (frontSum, dScores, sizeof(float) * colCount * rowCount, hipMemcpyDeviceToHost));
//		printf("FORNTSUM\n");
//		for(int i = 0; i < rowCount; i++)
//		{
//			for(int j = 0; j < colCount; j++)
//					{
//			printf(" %0.2f ", frontSum[j + i*colCount ]);
//		}
//		printf("\n");
//		}

	// <<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<

	RowSum(dScores,rowCount,colCount);

	// >>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>
	//int frontSum[colCount];
//	cutilSafeCall(hipMemcpy (frontSum, dScores, sizeof(float) * colCount * rowCount, hipMemcpyDeviceToHost));
//		printf("FORNTSUM\n");
//		for(int j = 0; j < rowCount; j++)
//		{
//			for(int i = 0; i < colCount; i++)
//				printf(" %0.2f ", frontSum[i+j*colCount]);
//			printf("\n");
//		}
//		printf("\n");

	// <<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<
	hipFree(dWorking);
	hipFree(dBaseSum);
}


__host__ int findFront( _teamReward* dData,int rowCount, int colCount, bool* dFront , int gapSize)
{


	bool* dTmpDistDom;
	bool* dTmpDistEqual;
	cutilSafeCall( hipMalloc( (void**) &dTmpDistDom,sizeof(bool) * rowCount *rowCount ));
	cutilSafeCall( hipMalloc( (void**) &dTmpDistEqual,sizeof(bool) *rowCount * rowCount  ));


	int* dTMP;
	cutilSafeCall( hipMalloc( (void**) &dTMP,sizeof(int) *rowCount * rowCount));


	_teamReward hData[rowCount*colCount];
	cutilSafeCall(hipMemcpy (hData, dData,sizeof(_teamReward) * rowCount*colCount , hipMemcpyDeviceToHost));
//	printf(">>>>DATA===\n");
//	for(int i=0;i < rowCount;i++)
//	{
//		for(int j=0;j < colCount;j++)
//			{
//				printf(" %d", hData[i*colCount+j]);
//			}
//		printf("\n");
//	}


//	int* hTMP = (int*)malloc(sizeof(int) * rowCount*rowCount);
//	for(int i=0; i < rowCount*rowCount; i++) { hTMP[i] =8;}
//	cutilSafeCall(hipMemcpy (dTMP, hTMP,sizeof(int) * rowCount*rowCount , hipMemcpyHostToDevice));


//	printf(" %d \n", rowCount);
	isDominated<<<dim3(rowCount,rowCount),dim3(128)>>>(dData,colCount,rowCount,dTmpDistDom,dTmpDistEqual, dTMP);
//	cutilSafeCall(hipMemcpy (hTMP, dTMP,sizeof(int) * rowCount*rowCount , hipMemcpyDeviceToHost));
//
//	printf("##$$##$$##$$\n");
//	for(int i=0;i < rowCount;i++)
//	{
//		for(int j=0;j < rowCount;j++)
//			{
//				printf(" %d", hTMP[i*rowCount+j]);
//			}
//		printf("\n");
//	}
//

	FindParetoFront(dTmpDistDom,dFront,rowCount);
	bool front[rowCount];
	cutilSafeCall(hipMemcpy (front, dFront,sizeof(bool) * rowCount , hipMemcpyDeviceToHost));

	int frontCount =0;
	for(int i=0; i< rowCount;i++)
		{
			if(front[i])
			{
				frontCount++;
			}
		}

	// >>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>.
	printf("selPoints F: ");
	for(int i=0; i< rowCount;i++)
	{
		if(front[i])
		{
			printf(" %d", i);
		//	frontCount++;
		}
	}
	printf("\nselPoints D: ");
	for(int i=0; i< rowCount;i++)
	{
		if(!front[i])
			printf(" %d", i);
	}
	printf("\n");
	// <<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<
	 hipFree( dTmpDistDom);
	 hipFree( dTmpDistEqual);

	 hipFree(dTMP);


		return frontCount;
}
__host__ void SelectTeamsGPU(int rowCount, int colCount, _teamReward* data, int gapSize, thrust::host_vector<int>* hToDel )
{

	printf(" Start: TSel\n");




	bool* dFront;
	cutilSafeCall( hipMalloc( (void**) &dFront,sizeof(bool) * rowCount ));

	_teamReward* dData;
	cutilSafeCall( hipMalloc( (void**) &dData,sizeof(_teamReward) * rowCount*colCount     ));
	cutilSafeCall(hipMemcpy (dData,data, sizeof(_teamReward) * rowCount*colCount   , hipMemcpyHostToDevice));


	bool front[rowCount];
	int frontCount = findFront(dData,rowCount,colCount,dFront, gapSize);
	cutilSafeCall(hipMemcpy (front,dFront, sizeof(bool) * rowCount   , hipMemcpyDeviceToHost));

	printf(" TF: ");
	for(int i=0;i<rowCount;i++)
		{
			if(front[i])
			{
				printf("    %d",i);
			}
		}
	printf("\n TD: ");
	for(int i=0;i<rowCount;i++)
		{
			if(!front[i])
			{
				printf("    %d",i);
			}
		}
	printf("\n\n");
	int keep = rowCount - gapSize;
	//printf("T:%d G:%d K:%d F:%d\n", rowCount,gapSize,keep,frontCount);
	if(frontCount == keep)
	{
		printf(" F == SAME\n");

		bool front[rowCount];
		cutilSafeCall(hipMemcpy (front,dFront, sizeof(bool) * rowCount   , hipMemcpyDeviceToHost));
		hToDel->clear();
		for(int i=0;i<rowCount;i++)
		{
			if(!front[i])
			{
				hToDel->push_back(i);
			}
		}
	//	frontToVector(front,rowSize,hToDel);
	}
	else{

		float* dScores;
		cutilSafeCall( hipMalloc( (void**) &dScores,sizeof(float) * rowCount*colCount ));

		if (frontCount < keep)
		{
			printf(" F == TOO SMALL\n");
			calcScores_D(dData,dFront, dScores,rowCount,colCount,frontCount);

			bool front[rowCount];
			cutilSafeCall(hipMemcpy (front, dFront,sizeof(bool) * rowCount , hipMemcpyDeviceToHost));

			float scores[rowCount*colCount];
			cutilSafeCall(hipMemcpy (scores, dScores,sizeof(float) * rowCount * colCount , hipMemcpyDeviceToHost));
			hToDel->clear();
			PTeamSelectParetoSerial(frontCount < keep, front,scores, rowCount,colCount, 0, gapSize,hToDel);
		}else{
			printf(" F == TOO BIG\n");
			calcScores_F(dData,dFront, dScores,rowCount,colCount,frontCount);

			bool front[rowCount];
					cutilSafeCall(hipMemcpy (front, dFront,sizeof(bool) * rowCount , hipMemcpyDeviceToHost));

					float scores[rowCount*colCount];
					cutilSafeCall(hipMemcpy (scores, dScores,sizeof(float) * rowCount * colCount , hipMemcpyDeviceToHost));

					hToDel->clear();
					for(int i=0;i < rowCount;i++)
															{
																if(!front[i]){
																	hToDel->push_back(i);
																}
															}
					PTeamSelectParetoSerial(frontCount < keep, front,scores, rowCount,colCount, 0,gapSize - (rowCount - frontCount),hToDel);



				//PselectParetoSerial(frontCount < keep, front,scores, rowCount, 0, gapSize,hToDel);
		}
		hipFree(dScores);
	}
	hipFree(dData);
	hipFree(dFront);


}

__host__ void SelectPointsGPU(int rowCount, int colCount, _teamReward* data, int gapSize, thrust::host_vector<_teamReward>* hToDel )
{
	int vecCount = colCount*colCount;
	_teamReward* dDist;

	//for(int i=0;i<20; i++) { data[i*rowCount] = i%10;}
//	printf(" BALLLLS\n");
//	for(int i =0; i < colCount;i++)
//	{
//		for(int j=0;j < rowCount;j++)
//		{
//			printf("%d,", data[i*rowCount+j]);
//		}
//	printf("\n");
//	}
//
//	printf("%d = %d * %d * %d\n" ,vecCount*rowCount , colCount,colCount,rowCount  );

	cutilSafeCall( hipMalloc( (void**) &dDist,sizeof(_teamReward) * vecCount*rowCount ));

	bool* dFront;
	cutilSafeCall( hipMalloc( (void**) &dFront,sizeof(bool) * rowCount ));

	_teamReward* dData;
	cutilSafeCall( hipMalloc( (void**) &dData,sizeof(_teamReward) * rowCount*colCount     ));
	cutilSafeCall(hipMemcpy (dData,data, sizeof(_teamReward) * rowCount*colCount   , hipMemcpyHostToDevice));

	GetDist(dData,dDist,colCount,rowCount);
//	kCalcDist<<<dim3(rowCount,1),dim3(512)>>>(dData, dDist, colCount, rowCount);
//	cutilCheckMsg("Kernel execution failed");

	//>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>
//	_teamReward* dist_out = (_teamReward*) malloc(sizeof(_teamReward) * vecCount*rowCount);
//	cutilSafeCall(hipMemcpy (dist_out,dDist, sizeof(_teamReward) *  vecCount*rowCount/*teamCount*teamCount*pointCount*/ , hipMemcpyDeviceToHost));
//	printf("selPoints distinctions\n");
//	for(int i = 0; i < rowCount;i++)
//	{
//		for(int j = 0; j < vecCount;j++)
//		{
//			printf("%d", dist_out[j+ i*vecCount]);
//			if(j%colCount == colCount-1){printf(" ");}
//		}
//		printf("\n");
//	}
//	printf("\n");

//	// <<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<


	int frontCount = findFront(dDist,rowCount,vecCount,dFront, gapSize);

	int keep = rowCount - gapSize;
	printf("T:%d G:%d K:%d F:%d\n", rowCount,gapSize,keep,frontCount);
	if(frontCount == keep)
	{
		printf(" F == SAME\n");

		bool front[rowCount];
		cutilSafeCall(hipMemcpy (front,dFront, sizeof(bool) * rowCount   , hipMemcpyDeviceToHost));
		hToDel->clear();
		for(int i=0;i<rowCount;i++)
		{
			if(!front[i])
			{
				hToDel->push_back(i);
			}
		}
	//	frontToVector(front,rowSize,hToDel);
	}
	else{

		float* dScores;
		cutilSafeCall( hipMalloc( (void**) &dScores,sizeof(float) * rowCount*colCount ));

		if (frontCount < keep)
		{
			printf(" F == TOO SMALL\n");
			calcScores_D(dData,dFront, dScores,rowCount,colCount,frontCount);

			bool front[rowCount];
			cutilSafeCall(hipMemcpy (front, dFront,sizeof(bool) * rowCount , hipMemcpyDeviceToHost));

			float scores[rowCount*colCount];
			cutilSafeCall(hipMemcpy (scores, dScores,sizeof(float) * rowCount * colCount , hipMemcpyDeviceToHost));
			hToDel->clear();
			PTeamSelectParetoSerial(frontCount < keep, front,scores, rowCount,colCount, 0, gapSize,hToDel);
		}else{
			calcScores_F(dData,dFront, dScores,rowCount,colCount,frontCount);

			bool front[rowCount];
					cutilSafeCall(hipMemcpy (front, dFront,sizeof(bool) * rowCount , hipMemcpyDeviceToHost));

					float scores[rowCount*colCount];
					cutilSafeCall(hipMemcpy (scores, dScores,sizeof(float) * rowCount * colCount , hipMemcpyDeviceToHost));
					hToDel->clear();
					for(int i=0;i < rowCount;i++)
										{
											if(!front[i]){
												hToDel->push_back(i);
											}
										}
					PTeamSelectParetoSerial(frontCount < keep, front,scores, rowCount,colCount, 0, gapSize,hToDel);


			//PselectParetoSerial(frontCount < keep, front,scores, rowCount, 0, gapSize,hToDel);
		}
		hipFree(dScores);
	}
	hipFree(dData);
	hipFree(dFront);
	hipFree(dDist);

	printf(" Leave: PSel\n");
}


//
//Hey JAzz
//-- Create TEstcases for TEamSelect
//-- Integrate PointSelect
//-- reduce code
//-- Pass Tests
//-- Clean once you think it works
//
//


