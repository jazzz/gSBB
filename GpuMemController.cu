#include "CudaControllerFunc.cuh"
#include "CudaControllerVars.cuh"

#define _GPU_MEM_VARS_

__host__
void initializeGpuEnv(int _maxLearnerCount , int _maxPointCount, int _maxProgSize,int _pointDim)
{
    numFeatures = _pointDim;
    learnerLength = _maxProgSize+1;

	maxLearnerCount = _maxLearnerCount;
	maxPointCount = _maxPointCount;

    bytesize_learner = sizeof(_learner ) * learnerLength;//maxProgSize;
    bytesize_singlePoint = sizeof(_point) * learnerLength;//maxProgSize;
    bytesize_learnerBid     = sizeof(_learnerBid)  * _maxPointCount;


}
//
//__host__ void destroyGpuEnv()
//{
//
//}
//__host__ void allocateDevicePointMatrix()
//{
//	cutilSafeCall( hipMalloc( (void**) &dPointMatrix, bytesize_singlePoint*maxPointCount));
//}
//__host__ void allocateDeviceBidMatrix()
//{
//	cutilSafeCall( hipMalloc( (void**) &dLearnerBidMatrix, bytesize_learnerBid*maxLearnerCount));
//}
//
//__host__ void freeDevicePointMatrix()
//{
//	cutilSafeCall( hipFree( dPointMatrix));
//}
//__host__ void freeDeviceBidMatrix()
//{
//	cutilSafeCall( hipFree( dLearnerBidMatrix));
//}
//
//
//
//__host__ void getDevicePointMatrix(_point* &ptr)
//{
//	ptr = dPointMatrix;
//}
//__host__ void getDeviceBidMatrix(_learnerBid* &ptr)
//{
//	ptr = dLearnerBidMatrix;
//}
//
//__host__ void replacePointInDeviceMatrix(int pointId, _point* hPoint)
//{
//	cutilSafeCall(hipMemcpy(&dPointMatrix[pointId*numFeatures], hPoint, bytesize_singlePoint*maxPointCount, hipMemcpyDeviceToHost));
//}
//
//__host__ void copyPointMatrixToHost(_point* hPoint)
//{
//	cutilSafeCall(hipMemcpy( hPoint, dPointMatrix, bytesize_singlePoint*maxPointCount, hipMemcpyDeviceToHost));
//}
//
//
//
//
//
