#include "hip/hip_runtime.h"
#define APP_DEBUG


#include "CudaControllerFunc.cuh"
#include "CudaControllerVars.cuh"

#include <thrust/device_vector.h>
#include "ErrorChecking.cuh"
#include "nvToolsExt.h"
#include "nvToolsExtCuda.h"
#include "nvToolsExtCudaRt.h"
#include "nvToolsExtMeta.h"

long Diff2(timeval tv_start, timeval tv_end){
   return 1000000*(tv_end.tv_sec - tv_start.tv_sec) + tv_end.tv_usec - tv_start.tv_usec;
}

#define WARP_SIZE 32
#define MODE_MASK 0x1
#define OP_MASK 0xE
#define DST_MASK 0x70
#define SRC_MASK 0x1F80

#define MODE_SHIFT 0
#define OP_SHIFT 1
#define DST_SHIFT 4
#define SRC_SHIFT 7

#define REG_COUNT 8

#define OP_SUM 0
#define OP_DIFF 1
#define OP_PROD 2
#define OP_DIV 3
#define OP_MOD 4
#define OP_COS 5
#define OP_EXP 6
#define OP_LOG 7
#define OP_CODE_COUNT 8


#define OpCodeCount 7
#define OP_NO 199
#define PIVOT_STRIPE_SIZE 16

__constant__ short NUM_FEATURES;
__constant__ short LEARNER_LENGTH;
__constant__ int TOTAL_LEARNERS;
__constant__ int TOTAL_POINTS;

#define KASSERT(condition)  if (condition) ;else{ return; }


typedef texture<int,2,hipReadModeElementType> tex;
tex texref;
__host__
void initializeGpuEnv(int _maxLearnerCount , int _maxPointCount, int _maxProgSize,int _pointDim)
{
    numFeatures = _pointDim;
    learnerLength = _maxProgSize+1;

	maxLearnerCount = _maxLearnerCount;
	maxPointCount = _maxPointCount;
	DEBUG_PRINT(("InitialVArs: LC:%d   MPC:%d    PS:%d   DIM:%d ",_maxLearnerCount,_maxPointCount,_maxProgSize,_pointDim ));
    bytesize_learner     = sizeof(_learner ) * learnerLength;//maxProgSize;
    bytesize_singlePoint = sizeof(_point) * numFeatures;//maxProgSize;
    bytesize_learnerBid  = sizeof(_learnerBid)  * _maxPointCount;


}
__host__ void setConstants(int totalPointCount, int totalLearnerCount, int learnerLength, int numFeatures)
{
	   CudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL( NUM_FEATURES), &numFeatures,sizeof(short)));
	   CudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL( LEARNER_LENGTH), &learnerLength,sizeof(short)));
	   CudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL( TOTAL_POINTS), &totalPointCount,sizeof(int)));

}


__host__ void destroyGpuEnv()
{

}

__host__ void startNVTXRange(char* str)
{
	nvtxRangePush(str);
//	      nvtxMark("Waiting...");
}
__host__ void stopNVTXRange(char* str)
{
	nvtxRangePop();
}

__host__ void allocateDevicePointMatrix()
{
	CudaSafeCall( hipMalloc( (void**) &dPointMatrix, bytesize_singlePoint*maxPointCount));
}
__host__ void allocateDeviceBidMatrix()
{
	CudaSafeCall( hipMalloc( (void**) &dLearnerBidMatrix, bytesize_learnerBid*maxLearnerCount));
}

__host__ void freeDevicePointMatrix()
{
	CudaSafeCall( hipFree( dPointMatrix));
}
__host__ void freeDeviceBidMatrix()
{
	CudaSafeCall( hipFree( dLearnerBidMatrix));
}



__host__ void getDevicePointMatrix(_point* &ptr)
{
	ptr = dPointMatrix;
}
__host__ void getDeviceBidMatrix(_learnerBid* &ptr)
{
	ptr = dLearnerBidMatrix;
}

__host__ void replacePointInDeviceMatrix(int pointId, _point* hPoint)
{
	CudaSafeCall(hipMemcpy(&dPointMatrix[pointId*numFeatures], hPoint, bytesize_singlePoint*maxPointCount, hipMemcpyDeviceToHost));
}

__host__ void copyPointMatrixToHost(_point* hPoint)
{
	CudaSafeCall(hipMemcpy( hPoint, dPointMatrix, bytesize_singlePoint*maxPointCount, hipMemcpyDeviceToHost));
}

__host__ void copyBidMatrixToHost(_learnerBid* hBid)
{
	CudaSafeCall(hipMemcpy( hBid, dLearnerBidMatrix, bytesize_learnerBid * maxLearnerCount, hipMemcpyDeviceToHost));
}

__host__ void pushPointMatrixToDevice(_point* hPoint, int size)
{
	_point* hPointPivotMatrix = (_point*) malloc( sizeof(_point) *size*numFeatures);
	int pivotStripeSize = PIVOT_STRIPE_SIZE;

	int numStripes = (size-1)/ pivotStripeSize+1;
	for(int pivotStripeId=0; pivotStripeId < numStripes; pivotStripeId++)
	{
		int offset = pivotStripeId*(pivotStripeSize * numFeatures);
		_point* pointPtr = &hPoint[offset];
		_point* pivotPtr = &hPointPivotMatrix[offset];

		//        for(int i = pivotStripeSize-1; i >0;--i)
		//        {
		//        	for(int j = numFeatures-1;j>0; --j)
		//        	{
		//        		if(i*numFeatures < pointCount)
		//        		pivotPtr[j*pivotStripeSize+i] = 2;// pointPtr[i*numFeatures+j];
		//        	}
		//        }
		for(int i = 0; i < pivotStripeSize; i++)
		{
			for(int j = 0;j < numFeatures; j++)
			{
				//if(i*numFeatures < pointCount)
				pivotPtr[j*pivotStripeSize+i] = pointPtr[i*numFeatures+j];
				//		printf("   %f",pointPtr[i*numFeatures+j] );
			}
			//printf("\n");
		}
	}
	CudaSafeCall(hipMemcpy(dPointMatrix, hPointPivotMatrix, bytesize_singlePoint * size, hipMemcpyHostToDevice));
	free(hPointPivotMatrix);
}
__host__ void pushBidMatrixToDevice(_learnerBid* hBid, int size)
{
	CudaSafeCall(hipMemcpy(dLearnerBidMatrix, hBid, bytesize_learnerBid * size, hipMemcpyHostToDevice));
}


__global__ void kLearnerEval(int learnOffset,
                      int pointOffset,
                      _learner* dLearnerMatrix,
                      _learnerBid* dLearnerBidMatrix,
                      _point* dPointMatrix,
                      int learnerCount,
                      int pointCount)
{

    int pointId = threadIdx.x + blockIdx.x * blockDim.x + pointOffset; // + (blockIdx.x*gridDim.x);
    int learnerId = threadIdx.y + blockIdx.y* blockDim.y + learnOffset;// + blockIdx.y * blockDim.y;
    if (learnerId < learnerCount && pointId < TOTAL_POINTS)
    {
		int id = threadIdx.x*blockDim.y + threadIdx.y;
		_learner *learner = &dLearnerMatrix[(threadIdx.y + blockIdx.y*blockDim.y) * LEARNER_LENGTH];
        _point *feature = &dPointMatrix[(threadIdx.x+blockDim.x*blockIdx.x)/PIVOT_STRIPE_SIZE * PIVOT_STRIPE_SIZE* NUM_FEATURES + ((threadIdx.x+blockDim.x*blockIdx.x)% PIVOT_STRIPE_SIZE) ];
      //_learner *shared_learner = &dLearnerMatrix[];
        //_point *feature = &dPointMatrix[0];

        KASSERT(LEARNER_LENGTH * blockDim.y < 16*50);
        __shared__ _learner shared_learner[16*50];
        int q=threadIdx.x;
             while(q < LEARNER_LENGTH)
             {
             	shared_learner[threadIdx.y*LEARNER_LENGTH+q] = learner[q];
             	q+= blockDim.x;
             }

        //     __syncthreads();
        _learnerBid registers[8];

        registers[0] =0;
        registers[1] =0;
        registers[2] =0;
        registers[3] =0;
        registers[4] =0;
        registers[5] =0;
        registers[6] =0;
        registers[7] =0;

        if(shared_learner[0] < 1){
     //   dLearnerBidMatrix[ learnerId*TOTAL_POINTS+ pointId] = -1 ;//learnerId+LEARNER_LENGTH ;
        }
        else{
        //short progsize = shared_learner[0];
        	int offset = threadIdx.y*LEARNER_LENGTH;
        for (int i=0;i<=shared_learner[0+offset];i++)
        {
            _learnerBid* dst = &registers[((shared_learner[i+offset] & DST_MASK) >> DST_SHIFT)];

            _learnerBid srcVal;

            if (1 == ((shared_learner[i+offset] & MODE_MASK) >> MODE_SHIFT ) %2) {

               // srcVal =  dPointMatrix[threadIdx.x* NUM_FEATURES + ((shared_learner[i] & SRC_MASK) >> SRC_SHIFT) % NUM_FEATURES ];
                srcVal =  feature[(((shared_learner[i+offset] & SRC_MASK) >> SRC_SHIFT) % NUM_FEATURES)*PIVOT_STRIPE_SIZE ];
            }else{
                srcVal =     registers[(((shared_learner[i+offset] & SRC_MASK) >> SRC_SHIFT) % REG_COUNT)];
            }
            switch ( ((shared_learner[i+offset] & OP_MASK) >> OP_SHIFT) % OP_CODE_COUNT){
            case OP_SUM:
                (*dst) += srcVal;
                break;
            case OP_DIFF:
                (*dst) -= srcVal;
                break;
            case OP_PROD:
                (*dst) *= srcVal;
                break;
            case OP_DIV:
                (*dst) /= srcVal;
                break;
            case OP_MOD:
                (*dst) =  fmod((*dst), srcVal);
                break;
            case OP_COS:
                (*dst) = cos(srcVal);
                break;
            case OP_EXP:
                (*dst) = expf(srcVal);
                break;
            case OP_LOG:
                (*dst) = logf(fabs(srcVal));
                break;
            }
            if(isfinite((*dst)) == 0)
                (*dst) = 0;


        }

       	dLearnerBidMatrix[ learnerId*TOTAL_POINTS + pointId] =   1 / (1+exp(-registers[0]));               //<<----------------------
       // dLearnerBidMatrix[ learnerId*TOTAL_POINTS + pointId] = (threadIdx.y < NUM_FEATURES) ? feature[threadIdx.y ] : -1;
		}
    }
}


__global__ void kTest(_learnerBid* it, _point* p, int learnerCount, int pointCount)
{
	  int pointId = threadIdx.x + blockIdx.x * blockDim.x ;
	  int learnerId = threadIdx.y + blockIdx.y* blockDim.y ;
	  if (learnerId < learnerCount && pointId < pointCount)
	  {

		  it[ learnerId*TOTAL_POINTS + pointId] =  pointId ;
	  }
}
__host__
void EvaluateLearners(_learner* hLearnerMatrix, _learnerBid* bidMatrix,_point* aaadPointMatrix, int learnerCount, int pointCount)
{


	int xThreads = 16;
	int yThreads = 16;

	int xBlocks = (pointCount-1)/xThreads+1;
	int yBlocks = (learnerCount-1)/yThreads+1;


    CudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL( TOTAL_LEARNERS), &learnerCount,sizeof(short)));
    CudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL( TOTAL_POINTS), &pointCount,sizeof(short)));


	_learnerBid* dA;
	CudaSafeCall( hipMalloc( (void**) &dA, bytesize_learner * learnerCount   ));
	CudaCheckError();
	_learner* dLearnerMatrix;
    CudaSafeCall( hipMalloc( (void**) &dLearnerMatrix, bytesize_learner * learnerCount   ));
	CudaSafeCall(hipMemcpy (dLearnerMatrix, hLearnerMatrix, bytesize_learner * learnerCount  , hipMemcpyHostToDevice));
	CudaSafeCall(hipMemset(dA,1,40));
//	printf("XB:%d  YB:%d  xT:%d  yT:%d  LC:%d  PC:%d\n",xBlocks,yBlocks,xThreads,yThreads,learnerCount,pointCount);
	//kTest<<<dim3(xBlocks,yBlocks),dim3(xThreads,yThreads)>>>(dLearnerBidMatrix,dPointMatrix, learnerCount,pointCount);
	kLearnerEval<<<dim3(xBlocks,yBlocks),dim3(xThreads,yThreads)>>>(0,0,dLearnerMatrix,dLearnerBidMatrix,dPointMatrix,learnerCount, pointCount);
	CudaCheckError();
	//CudaSafeCall(hipMemcpy (bidMatrix, dA, bytesize_learner * learnerCount   , hipMemcpyDeviceToHost));
	//copyBidMatrixToHost(bidMatrix);
//	_learnerBid A[5];
//	A[0] = 5;
//	A[1] = 5;
//	A[2] = 5;
//	A[3] = 5;
//	A[4] = 5;
	//aCudaSafeCall(hipMemcpy (dLearnerBidMatrix, &A, 5  , hipMemcpyHostToDevice));

	//bidMatrix[5] = 2;
	hipFree(dLearnerMatrix);
	hipFree(dA);
}



__host__ void
cLearnerEvalSingle(_learner* learner, _learnerBid &learnerBid, _point* feature, int LEARNER_LENGTH, int NUM_FEATURES )
{

    // printf("cLEArn SINGLE\n");


    float testVal =-1;

    _learnerBid registers[REG_COUNT] ={0};

//		printf("FEATURE ");
//	for(int _a=0;_a< NUM_FEATURES;_a++)
//	{
//		printf(" %f",feature[_a]);
//	}
//		printf("\n");

    short progsize = learner[0];
//     printf(" PROGSIZE: %d" , progsize);
    short val = 0;
    for (int i=1;i<progsize+1;i++)
    {
        unsigned short instruction = learner[i] ;//& 8191;
              // printf( " INST:%d ", instruction);

//		unsigned short inst = instruction;
//		char str[14];
//		str[13] = '\0';
//		for(int p =0 ; p < 13; p++)
//		{
//			str[12-p] = (inst%2==0) ? '0':'1';
//			inst /=2;
//		}
//		printf("%s",str);


        unsigned char mode =0;
        mode = ((instruction & MODE_MASK) >> MODE_SHIFT ) %2;
        char op = 0;
        op = ((instruction & OP_MASK) >> OP_SHIFT) % OP_CODE_COUNT;
        short dst_index = (instruction & DST_MASK) >> DST_SHIFT ; //% REG_COUNT ;
//          printf(" R[%d] <- R[%d] ", dst_index, dst_index);
        _learnerBid* dst = &registers[(instruction & DST_MASK) >> DST_SHIFT];
        unsigned short src = (instruction & SRC_MASK) >> SRC_SHIFT;
        //  float srcVal =	(mode == 1) ? feature[src%NUM_FEATURES] : registers[src%REG_COUNT]   ;
        _learnerBid srcVal;
        // printf(" Mode:%d R[%d]",mode, dst_index );


        if ( mode ==1) {
            srcVal = feature[src%NUM_FEATURES];
//               printf("I[%d] ", src%NUM_FEATURES);
        }else{
            srcVal =     registers[src%REG_COUNT];
//              printf("R[%d] ", src%REG_COUNT);
        }

        //printf(" OP:%d (", op);
        switch (op){
        case OP_SUM:
            (*dst) += srcVal;
//             printf("sum ");
            testVal=srcVal;
            break;
        case OP_DIFF:
            (*dst) -= srcVal;
//                 printf("diff ");
            testVal=1;
            break;
        case OP_PROD:
            (*dst) *= srcVal;
//               printf("prod ");
            testVal=2;
            break;
        case OP_DIV:
            (*dst) /= srcVal;
//               printf("div ");
            testVal=3;
            break;
        case OP_MOD:
            (*dst) =  fmod((*dst), srcVal);
//              printf("mod ");
            testVal=4;
            break;
        case OP_COS:
            (*dst) = cos(srcVal);
//                    printf("cos ");
            break;
        case OP_EXP:
        	//if(srcVal > 88 && srcVal < 709.783){(*dst)=FLT_MAX;}										// Floats Wrapping with EXP -- easiest way to stop it
        	(*dst) = exp(srcVal);
//                   printf("exp ");

            break;
        case OP_LOG:
            (*dst) = logf(fabs(srcVal));
//                printf("log ");
            break;
        }
        if(isfinite((*dst)) == 0)
        {
//        	if( isinf((*dst)) )
//        	{
//        		(*dst) = FLT_MAX;
//        	}
//        	//printf(" DSTNOTFINITE: %f", (*dst) );
//        	else{
            (*dst) = 0;
 //       	}
        }
//                printf("REG ");
//                for ( int i = 0; i < REG_COUNT; i++)
//                {
//
//                    printf("%f ", registers[i]);
//                }
//                printf("\n");
    }
    //        printf(" final:: ");
    //      for ( int i = 0; i < REG_COUNT; i++)
    //      {
    //          printf("   %f ", registers[i]);
    //      }

//    printf(":%f", 1 / float(1+exp(-registers[0])));
    learnerBid = 1 / float(1+exp(-registers[0]));
}



#define BIGNUMBER 99999;
#define STATE_FRONT_TOO_SMALL 0
#define STATE_FRONT_TOO_BIG 1
#define STATE_FRONT_IS_JUUUST_RIGHT 2

int MSB(int v)
{
const unsigned int b[] = {0x2, 0xC, 0xF0, 0xFF00, 0xFFFF0000};
const unsigned int S[] = {1, 2, 4, 8, 16};
int i;

register unsigned int r = 0; // result of log2(v) will go here
for (i = 4; i >= 0; i--) // unroll for speed...
{
  if (v & b[i])
  {
    v >>= S[i];
    r |= S[i];
  }
}
return r;
}

int nextPowerOf2(int v)
{
	return 1<<(MSB(v)+1);
}


__global__ void kCalcDist(_teamReward* rewards, _teamReward* dist_out, int teamCount, int pointCount)
{
	if(threadIdx.x < teamCount && blockIdx.x < pointCount)
	{
		_teamReward* dist;
	//	_teamReward* rewards;
		int j = threadIdx.x;

		dist = dist_out +  blockIdx.x*teamCount*teamCount;

		for(int i=0;i < teamCount; i++)
		{
				dist[j+i*pointCount] = (rewards[i*teamCount+blockIdx.x] > rewards[j*teamCount+blockIdx.x]);

			//	if(j==1){dist[j+i*teamCount] = 1;}
		}

	}
}
// lets assume 16x16 for fun
__global__ void kCalcDistSoMuchBetterBro__(_teamReward* rewards, _teamReward* dist_out, int teamCount, int pointCount)
{
	int pointIndex = blockIdx.z;
	int i = blockIdx.y*blockDim.y + threadIdx.y;
	int j = blockIdx.x*blockDim.x + threadIdx.x;

//	extern __shared__ _teamReward XCache[];					GAH You 2 hours debug later: TL;DR can't have two separate arrays
//	extern __shared__ _teamReward YCache[];
	extern __shared__ _teamReward Cache[];
	if(i < teamCount && j < teamCount && pointIndex < pointCount)
	{

			if(0 == threadIdx.y )
			{
				Cache[threadIdx.x] = rewards[j*pointCount+pointIndex];
			}
			if(0 == threadIdx.x )
			{
				Cache[threadIdx.y+blockDim.x] = rewards[i*pointCount+pointIndex];
			}

	}
		__syncthreads();
		if(i < teamCount && j < teamCount && pointIndex < pointCount)
		{
			dist_out[pointIndex*teamCount*teamCount + i*teamCount+j] =  Cache[threadIdx.y+blockDim.x] > Cache[threadIdx.x];
		}





}

// lets assume 16x16 for fun
__global__ void kCalcDistSoMuchBetterBro(_teamReward* rewards, _teamReward* dist_out, int teamCount, int pointCount)
{
//	int pointIndex = blockIdx.z;
//	int i = blockIdx.y*blockDim.y + threadIdx.y;
//	int j = blockIdx.x*blockDim.x + threadIdx.x;

		if(threadIdx.x < teamCount && threadIdx.y < teamCount)
		{

			dist_out[threadIdx.y*blockDim.x+threadIdx.x] = rewards[threadIdx.y+pointCount*blockIdx.z] > rewards[threadIdx.x+pointCount*blockIdx.z];
//			dist_out[pointIndex*teamCount*teamCount + j*teamCount+i] = 3;
//			dist_out[pointIndex*teamCount*teamCount + j] = 3;
		}
//




}

__global__ void kCalcDistTex(_teamReward* dist_out, int teamCount, int pointCount)
{
	int pointIndex = blockIdx.z;
	int i = blockIdx.y*blockDim.y + threadIdx.y;
	int j = blockIdx.x*blockDim.x + threadIdx.x;

		if(i < teamCount && j < teamCount && pointIndex < pointCount)
		{

			dist_out[pointIndex*teamCount*teamCount + i*teamCount+j] = tex2D(texref,j,pointIndex) < tex2D(texref,i,pointIndex) ;
		}
//




}

__host__ void GetDist(_teamReward* rewards, _teamReward* dist_out, int teamCount, int pointCount)
{
	//int threads = 16;
	//int xThreads = threads;
	//int yThreads = threads;
	int xThreads = 16;
	int yThreads = 16;


	int xBlocks = (teamCount-1) / xThreads +1;
	int yBlocks = (teamCount-1) / yThreads +1;
	int zBlocks = pointCount;


	//hipMemset(dist_out,0,sizeof(_teamReward) * teamCount*teamCount*pointCount);


	if(xThreads*yThreads > 1024){ fprintf(stderr,"Error: Too many threads used in GetDist");}
	//kCalcDistSoMuchBetterBro<<<dim3(xBlocks,yBlocks,zBlocks),dim3(xThreads),xThreads>>>(rewards, dist_out, teamCount, pointCount);
	kCalcDistSoMuchBetterBro__<<<dim3(xBlocks,yBlocks,zBlocks),dim3(xThreads,yThreads),xThreads+yThreads>>>(rewards, dist_out, teamCount, pointCount);
	CudaCheckError();

}

__host__ void GetDistTex(_teamReward* rewards, _teamReward* dist_out, int teamCount, int pointCount)
{
	//int threads = 16;
	//int xThreads = threads;
	//int yThreads = threads;
	int xThreads = 16;
	int yThreads = 16;


	int xBlocks = (teamCount-1) / xThreads +1;
	int yBlocks = (teamCount-1) / yThreads +1;
	int zBlocks = pointCount;


	//hipMemset(dist_out,0,sizeof(_teamReward) * teamCount*teamCount*pointCount);

	hipArray* carray;
	hipChannelFormatDesc channel;

	channel = hipCreateChannelDesc<int>();

	int rowCount = teamCount;
	int colCount = pointCount;
	hipMallocArray(&carray,&channel,rowCount, colCount);
	hipMemcpyToArray(carray,0,0,rewards,sizeof(int)* rowCount*colCount, hipMemcpyHostToDevice);

	texref.filterMode=hipFilterModePoint;
	texref.addressMode[0]=hipAddressModeClamp;
	texref.addressMode[1]=hipAddressModeClamp;
	hipBindTextureToArray(texref,carray);
	if(xThreads*yThreads > 1024){ fprintf(stderr,"Error: Too many threads used in GetDist");}
	kCalcDistTex<<<dim3(xBlocks,yBlocks,zBlocks),dim3(xThreads,yThreads),xThreads+yThreads>>>(dist_out, teamCount, pointCount);
	CudaCheckError();

	hipUnbindTexture(texref);
	hipFree(carray);

}


__global__ void isDominated___(_teamReward* Vec, int size, int vecCount,bool* DOM, bool* EQUAL, int* OUT)
{

	int isIndex = blockIdx.x;
	int byIndex = blockIdx.y;


		_teamReward* vIs= &Vec[size*isIndex];
		_teamReward* vBy= &Vec[size*byIndex];
		__shared__ bool dominated[128];
		__shared__ bool equal[128];

		__shared__ bool isDominated;
		__shared__ bool isEqual;
		if(threadIdx.x ==0){isDominated = true; isEqual =true;}
		__syncthreads();

		int elementOffset = 0;
		while(isDominated && elementOffset*blockDim.x < size)
		{
			int tid = 0;
			tid = threadIdx.x + elementOffset;


			//int active_threads = (size - stepIndex*blockDim.x > blockDim.x) ? blockDim.x : size ;
			if(tid < size)
			{
				dominated[threadIdx.x] = (vIs[tid] > vBy[tid]);
				equal[threadIdx.x] = (vIs[tid] == vBy[tid]);
			}else{
				dominated[threadIdx.x] = 0;
				equal[threadIdx.x] = 1;
			}

			__syncthreads();

			int activeThreads = blockDim.x/2;

			while(activeThreads >0)
			{
				if(threadIdx.x < activeThreads && tid+activeThreads < size)
				{
					dominated[threadIdx.x] = dominated[threadIdx.x] || dominated[threadIdx.x+activeThreads];
					equal[threadIdx.x] = equal[threadIdx.x] && equal[threadIdx.x+activeThreads];
				}

				__syncthreads();
				activeThreads /=2;
			}

		__syncthreads();
			if(threadIdx.x ==0)
			{
				isDominated = !dominated[0];
				isEqual = isEqual && equal[0];
					//DOM[isIndex*blockDim.y + byIndex] = dominates[0];
					//EQUAL[isIndex*blockDim.y + byIndex] = equal[0];
			}

			elementOffset += blockDim.x;



		}

		__syncthreads();

		if(threadIdx.x ==0)
		{

			//EQUAL[isIndex*vecCount + byIndex] = isEqual;
			DOM[isIndex*vecCount + byIndex] = (byIndex < isIndex && isEqual) || (isDominated && !isEqual);
			OUT[isIndex*vecCount + byIndex] =  dominated[0];
		}



}
__global__ void isDominated_single(_teamReward* Vec, int size, int vecCount,bool* DOM, bool* EQUAL, int* OUT)
{

	int isIndex = blockIdx.x;
	int byIndex = blockIdx.y;
	int offset = blockIdx.z * threadIdx.x;


	_teamReward* vIs= &Vec[size*isIndex];
	_teamReward* vBy= &Vec[size*byIndex];
	__shared__ bool dominated[128];
	__shared__ bool equal[128];

}


__global__ void isDominated(_teamReward* Vec, int size, int vecCount,bool* DOM, bool* EQUAL, int* OUT)
{

	int isIndex = blockIdx.x;
	int byIndex = blockIdx.y;


		_teamReward* vIs= &Vec[size*isIndex];
		_teamReward* vBy= &Vec[size*byIndex];
		__shared__ bool dominated[128];
		__shared__ bool equal[128];

		__shared__ bool isDominated;
		__shared__ bool isEqual;
		if(threadIdx.x ==0){isDominated = true; isEqual =true;}
		__syncthreads();

		int elementOffset = 0;
		while(isDominated && elementOffset*blockDim.x < size)
		{
			int tid = 0;
			tid = threadIdx.x + elementOffset;


			//int active_threads = (size - stepIndex*blockDim.x > blockDim.x) ? blockDim.x : size ;
			if(tid < size)
			{
				dominated[threadIdx.x] = (vIs[tid] > vBy[tid]);
				equal[threadIdx.x] = (vIs[tid] == vBy[tid]);
			}else{
				dominated[threadIdx.x] = 0;
				equal[threadIdx.x] = 1;
			}

			__syncthreads();

			int activeThreads = blockDim.x/2;

			while(activeThreads >0)
			{
				if(threadIdx.x < activeThreads && tid+activeThreads < size)
				{
					dominated[threadIdx.x] = dominated[threadIdx.x] || dominated[threadIdx.x+activeThreads];
					equal[threadIdx.x] = equal[threadIdx.x] && equal[threadIdx.x+activeThreads];
				}

				__syncthreads();
				activeThreads /=2;
			}

		__syncthreads();
			if(threadIdx.x ==0)
			{
				isDominated = !dominated[0];
				isEqual = isEqual && equal[0];
					//DOM[isIndex*blockDim.y + byIndex] = dominates[0];
					//EQUAL[isIndex*blockDim.y + byIndex] = equal[0];
			}

			elementOffset += blockDim.x;



		}

		__syncthreads();

		if(threadIdx.x ==0)
		{

			//EQUAL[isIndex*vecCount + byIndex] = isEqual;
			DOM[isIndex*vecCount + byIndex] = (byIndex < isIndex && isEqual) || (isDominated && !isEqual);
			OUT[isIndex*vecCount + byIndex] =  isDominated;
		}



}


// TODO: BRO fix  this -- Gonn ahve to loop over an do a recursive reduction
__global__ void findParetoFront(bool* DOM, bool* front, int size)
{
	int pointId = blockIdx.y + blockIdx.x*gridDim.x;
	int index = threadIdx.x;


	extern __shared__ bool data[];



	int offset = blockDim.x;
	data[threadIdx.x] = ((index < size) ? DOM[pointId*size+index]: 0) or ((index + offset < size) ? DOM[pointId*size + index + offset] : 0);



	// Reduce
	__syncthreads();
	int activeThreads = blockDim.x/2;
	while(activeThreads > 0)
	{
		if(threadIdx.x < activeThreads)
			data[threadIdx.x] = data[threadIdx.x] or data[threadIdx.x + activeThreads];
		__syncthreads();
		activeThreads /= 2;
	}

	__syncthreads();
	if(0 == threadIdx.x )
		front[pointId] = data[0];


}

__global__ void findParetoFront2(bool* DOM, bool* out,int size, int pointOffset, int* out2)
{
	int rowId = blockIdx.y ;//+ pointOffset;
	int index = threadIdx.x ;//+ blockIdx.x*blockDim.x;


	extern __shared__ bool data[];


	int offset = blockDim.x;
	data[threadIdx.x] = ((index < size) ? DOM[rowId*size+index]: 0) or ((index + offset < size) ? DOM[rowId*size + index + offset] : 0);


	// Reduce
	__syncthreads();
	int activeThreads = blockDim.x/2;
	while(activeThreads > 0)
	{
		if(threadIdx.x < activeThreads)
			data[threadIdx.x] = data[threadIdx.x] or data[threadIdx.x + activeThreads];
		__syncthreads();
		activeThreads /= 2;
	}

	__syncthreads();
	if(0 == threadIdx.x )
	{
		out[rowId*gridDim.x+blockIdx.x] = !data[0];
	}



}
__global__ void col2Row(bool* to, bool* from,int rowCount,int colCount)
{
	int a;
	if( threadIdx.x < rowCount)
	{
		to[threadIdx.x] = from[threadIdx.x*colCount];
	}

}

__host__ void FindParetoFront(bool* dom, bool* front,int size)	// PS its  rowSize * rowSize Matrix
{
	int rowsPerInvocation = size;

	int colCount = size;
	int rowCount = size;

	int xThreads = 128;
	int xBlocks = (colCount-1) / xThreads +1;
	int yBlocks = rowCount;


	CudaCheckError();
//	bool* A = new bool[rowCount*colCount];
//	CudaSafeCall(hipMemcpy (A, dom, sizeof(bool) * colCount * rowCount, hipMemcpyDeviceToHost));
//
//	printf("====^^^^^====\n");
//		for(int i =0; i < rowCount; i++)
//		{
//			for(int j=0;j<colCount;j++)
//			{
//				printf(" %s" , (A[i*colCount+j]) ? "1":"0" );
//			}
//			printf("\n");
//		}

		int colsLeft = colCount;



	bool* dStaging;
	CudaSafeCall( hipMalloc( (void**) &dStaging,sizeof(bool) * rowCount * xBlocks   ));
	int* tmp;
		CudaSafeCall( hipMalloc( (void**) &tmp,sizeof(int) * rowCount * rowCount   ));
	//	CudaSafeCall(hipMemcpy (dWorking, dData, sizeof(int) * colCount *rowCount, hipMemcpyDeviceToDevice));

	int rowOffset = 0;

						// TODO: Which way? X blocks First or Y Block First (X = finnish Summation? Y =Finish all rows first
	while(colsLeft > 1)
	{
		xBlocks = (colsLeft-1)/xThreads+1;

		while(rowOffset < yBlocks)
		{

	//		printf("BSS %d\n", xBlocks);
			int blocksPerGrid_x =  xBlocks;
			int blocksPerGrid_y = (rowCount > rowsPerInvocation) ? rowsPerInvocation : rowCount;

			findParetoFront2<<<dim3(1,blocksPerGrid_y),xThreads,xThreads>>>(dom,dStaging,colsLeft,rowOffset,tmp);
			CudaCheckError();
			rowOffset += blocksPerGrid_y;
		}
		colsLeft = xBlocks;
	}
		int t[rowCount*rowCount];
//		CudaSafeCall(hipMemcpy (t, tmp, sizeof(int) * rowCount *rowCount, hipMemcpyDeviceToHost));
//		CudaSafeCall(hipMemcpy (A, dStaging, sizeof(bool) * xBlocks *rowCount, hipMemcpyDeviceToHost));
//		printf("====TMP====");
//		for(int i =0; i < rowCount; i++)
//				{
//					for(int j=0;j< rowCount;j++)
//					{
//						printf(" %d" , t[i*rowCount+j]);
//					}
//					printf("\n");
//				}
//		printf("====#====");
//		for(int i =0; i < rowCount; i++)
//		{
//			for(int j=0;j< xBlocks;j++)
//			{
//				printf(" %s" , (A[i*xBlocks+j]) ? "1":"0" );
//			}
//			printf("\n");
//		}
//
//	printf(" R == %d\n" ,nextPowerOf2(1204));
	col2Row<<<1,nextPowerOf2(rowCount)>>>(front,dStaging,rowCount,xBlocks);					// LIMIT: rowCount == 1024
	CudaCheckError();

	//	CudaSafeCall(hipMemcpy (front, dStaging, sizeof(bool) *rowCount, hipMemcpyDeviceToDevice));

//	CudaSafeCall(hipMemcpy (A, front, sizeof(bool) * rowCount, hipMemcpyDeviceToHost));
//		printf("========");
//		for(int i =0; i < rowCount; i++)
//		{
////			for(int j=0;j< xBlocks;j++)
////			{
//				printf(" %s" , (A[i]) ? "1":"0" );
////			}
//			printf("\n");
//		}
//
//

//	delete A;
	hipFree(tmp);
	hipFree(dStaging);


}
__global__ void kColumnSum(int* vec, int vecCount, int rowCount,int* out, int offset)
{
	int vecIndex = blockIdx.x + offset;
	int rowIndex = threadIdx.y + blockIdx.y*blockDim.y;
	__shared__ int sum_vec[256];

	sum_vec[threadIdx.y] = 0;

	if(rowIndex < rowCount)
	{
		sum_vec[threadIdx.y] = vec[vecIndex+ rowIndex*vecCount]; //vec[vecIndex + pointCount*vecCount ];
	}



int i = blockDim.y;
i /=2 ;
while(i>0)
{
	__syncthreads();
	if( threadIdx.y < i  )
	{
		sum_vec[threadIdx.y] = sum_vec[threadIdx.y]+ sum_vec[threadIdx.y+ i];//threadIdx.y* 1000 + threadIdx.y+(i/2);

	}
	i /=2 ;

}


	__syncthreads();

	if(threadIdx.y==0)
	{
		out[vecIndex + blockIdx.y*vecCount] = sum_vec[0];

	}


}

__host__ void ColumnSum(int* vec, int rowCount, int colCount)
{
	int yThreads = 128;
	int yBlocks = (rowCount -1)/ yThreads +1;
	CudaCheckError();
	int blocksPerGrid_x = (colCount > 40) ? 40 : colCount;
	for(int _rowsLeft=rowCount; _rowsLeft >1;_rowsLeft = (_rowsLeft -1) / yThreads+1)
	{
		int elementsRemaining = colCount;
		for(int offset = 0; offset+blocksPerGrid_x <= colCount; offset += blocksPerGrid_x )
		{
			kColumnSum <<<dim3(blocksPerGrid_x,yBlocks),dim3(1,yThreads)>>>(vec,colCount, rowCount, vec,offset);
			CudaCheckError();
		}

	}
}

template< typename T >
__global__ void kRowSum(T* vec, int rowCount, int vecCount,T* out, int offset)
{
	int vecIndex = threadIdx.x + blockIdx.x*blockDim.x;
	int rowIndex = blockIdx.y + offset;
	__shared__ T sum_vec[256];

	sum_vec[threadIdx.x] = 0;

//	if(rowIndex < rowCount)
//	{
//		sum_vec[threadIdx.x] = vec[vecIndex+ rowIndex*vecCount] ; //vec[vecIndex + pointCount*vecCount ];
//	}
	sum_vec[threadIdx.x] = ((vecIndex < vecCount) ?  vec[vecIndex+ rowIndex*vecCount]: 0); //or ((index + offset < size) ? DOM[pointId*size + index + offset] : 0);


	int i = blockDim.x/2;
	while(i > 0)
	{
		__syncthreads();
		if(threadIdx.x <i)
			sum_vec[threadIdx.x] = sum_vec[threadIdx.x]+ sum_vec[threadIdx.x+ i];//threadIdx.y* 1000 + threadIdx.y+(i/2);
		i /=2;
	}



	__syncthreads();

	if(threadIdx.x  < vecCount)
	{
		out[vecIndex + rowIndex*vecCount] = sum_vec[threadIdx.x];

	}


}
__host__ void RowSum(float* dVec, int rowCount, int colCount)
{
	int xThreads = 128;
	int xBlocks =  (colCount -1)/ xThreads +1;
	int blocksPerGrid_y = (rowCount > 40) ? 40 : rowCount;
	for(int _rowsLeft=colCount; _rowsLeft >1;_rowsLeft = (_rowsLeft -1) / xThreads+1)
	{
		int elementsRemaining = colCount;
		for(int offset = 0; offset+blocksPerGrid_y <= rowCount; offset += blocksPerGrid_y )
		{
			kRowSum <<<dim3(xBlocks,blocksPerGrid_y),dim3(xThreads,1)>>>(dVec,rowCount, colCount, dVec,offset);

			CudaCheckError();
		}

	}
}
template< typename T >
__global__ void FrontFilter(T* vec, bool* front, bool Bool, T val, int vecCount, int pointCount, T* out,int offset)
{
	int vecIndex = blockIdx.x + offset;
	int pointIndex = threadIdx.y + blockIdx.y*blockDim.y;

	if(pointIndex < pointCount )
	{
		if(Bool != front[pointIndex])
		{
			out[vecIndex + pointIndex * vecCount] = val;
		}
		//out[vecIndex + pointIndex * vecCount] = (Bool == front[pointIndex]) ? vec[vecIndex + pointCount*vecCount ] : 0;
	}

}

void PointSelectParetoSerial(bool mode, bool* front, float* scores, int rowCount, int colCount,int _frontCount, int nrem, thrust::host_vector<_teamReward>* toDel)
{

	  struct timeval tv_1;
		struct timeval tv_2;
		struct timezone tz;
		long timer0 = 0;
		gettimeofday(&tv_1, &tz);

	//Sort
	thrust::host_vector<int> hIndex(0);
	//thrust::sequence(hIndex.begin(), hIndex.end());
	thrust::host_vector<float> hScores(0);


	thrust::host_vector<int> hFrontMask(rowCount);
	thrust::device_vector<int> dFrontMask(rowCount);
	int frontCount =0;
	//printf("IDESCAORES: ");
	for(int i=0;i<rowCount;i++)
	{
		if(front[i]== !mode)
		{
			frontCount++;
			hIndex.push_back(i);
			hScores.push_back(scores[i*colCount]);
	//		printf(" %f", scores[i*colCount]);
		//	hFrontMask.push_back(front[i]);
		}

	}
//
//	printf("\nNot Sorted  ");
//		for(int i =0; i <hIndex.size();i++)
//			{
//				printf("  $%d::%f ",hIndex[i], hScores[i]);
//			}
//		printf("\n");
	for(int i= hIndex.size()-1; i > 0 ;i--)
	{
		for(int j=0; j < i;j++)
		{
			if(hScores[j] > hScores[j+1])
			{
				int a = hIndex[j];
				float b = hScores[j];

				hIndex[j] = hIndex[j+1];
				hScores[j] = hScores[j+1] ;

				hIndex[j+1] = a;
				hScores[j+1] = b ;
			}
		}
	}
//printf("\nSoreted  ");
//	for(int i =0; i <hIndex.size();i++)
//		{
//			printf("  $%d::%f ",hIndex[i], hScores[i]);
//		}
//	printf("\n");

//	printf("DELETE");
	for(int i=0; i < nrem;i++)
	{
//		printf("   %d", hIndex[i]);
		toDel->push_back(hIndex[i]);
	}
//	printf("\n");

	//dFrontMask = hFrontMask;
	//int result = thrust::count(dFrontMask.begin(), dFrontMask.end(), 1);

	//printf("FrontCount = %d vs %d",count,result );


//	for(int i=0; i < pointCount;i++)
//	{
//		printf("  %d:%f", i,scores[i]);
//	}
//	printf("\n");




	// Find NLowest
	gettimeofday(&tv_2, &tz);
		timer0 = Diff2(tv_1,tv_2);
//		printf(" Stage Pareto  : %ld\n", timer0 );
}

void PTeamSelectParetoSerial(bool mode, bool* front, float* scores, int rowCount, int colCount,int _frontCount, int nrem, thrust::host_vector<_teamReward>* toDel)
{

	  struct timeval tv_1;
		struct timeval tv_2;
		struct timezone tz;
		long timer0 = 0;
		gettimeofday(&tv_1, &tz);

	//Sort
	thrust::host_vector<int> hIndex(0);
	//thrust::sequence(hIndex.begin(), hIndex.end());
	thrust::host_vector<float> hScores(0);


	thrust::host_vector<int> hFrontMask(rowCount);
	thrust::device_vector<int> dFrontMask(rowCount);
	int frontCount =0;
//	printf("IDESCAORES: ");
	for(int i=0;i<rowCount;i++)
	{
		if(front[i]== !mode)
		{
			frontCount++;
			hIndex.push_back(i);
			hScores.push_back(scores[i*colCount]);
//			printf(" %f", scores[i*colCount]);
		//	hFrontMask.push_back(front[i]);
		}

	}
//
	printf("\nNot Sorted  ");
		for(int i =0; i <hIndex.size();i++)
			{
				printf("  $%d::%f ",hIndex[i], hScores[i]);
			}
		printf("\n");
	for(int i= hIndex.size()-1; i > 0 ;i--)
	{
		for(int j=0; j < i;j++)
		{
			if(hScores[j] > hScores[j+1])
			{
				int a = hIndex[j];
				float b = hScores[j];

				hIndex[j] = hIndex[j+1];
				hScores[j] = hScores[j+1] ;

				hIndex[j+1] = a;
				hScores[j+1] = b ;
			}
		}
	}
printf("\nSoreted  ");
	for(int i =0; i <hIndex.size();i++)
		{
			printf("  $%d::%f ",hIndex[i], hScores[i]);
		}
	printf("\n");

	printf("DELETE");
	for(int i=0; i < nrem;i++)
	{
		printf("   %d", hIndex[i]);
		toDel->push_back(hIndex[i]);
	}
	printf("\n");

	//dFrontMask = hFrontMask;
	//int result = thrust::count(dFrontMask.begin(), dFrontMask.end(), 1);

	//printf("FrontCount = %d vs %d",count,result );


//	for(int i=0; i < pointCount;i++)
//	{
//		printf("  %d:%f", i,scores[i]);
//	}
//	printf("\n");




	// Find NLowest
	gettimeofday(&tv_2, &tz);
		timer0 = Diff2(tv_1,tv_2);
//		printf(" Stage Pareto  : %ld\n", timer0 );
}
__global__ void kElementDivide(int* A, int* B, int vecCount, float* out)
{
	if(threadIdx.x + blockIdx.x*blockDim.x < vecCount)
	out[threadIdx.x + blockIdx.x*blockDim.x] = (A[threadIdx.x +blockIdx.x*blockDim.x]+1) /  (float(B[threadIdx.x + blockIdx.x*blockDim.x])+1);
}

__global__ void kColumnNormalize(int* dData,int* dBaseSum,int rowCount, int colCount, float* dOut)
{
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	extern __shared__ int divisor[];

	if(row< rowCount && col < colCount)
	{
	if(threadIdx.y == 0)
	{
		divisor[threadIdx.x] = dBaseSum[col];
	}
	}
	__syncthreads();
	if(row< rowCount && col < colCount)
		{
	dOut[row*colCount+col] = float(dData[row*colCount+col])/divisor[threadIdx.x];
	if(false == isfinite(dOut[row*colCount+col])){ dOut[row*colCount+col] = 0;}
		}

}


__host__ void frontToVector(bool* front,int rowCount,thrust::host_vector<int>* hToDel  )
{
	int count =0;
	for(int i=0;i<rowCount;i++){
		if(!front[i]){
			(*hToDel)[count++]=i;
		}
	}
}

__host__ void calcScores_D(_teamReward* dData, bool* dFront, float* dScores, int rowCount, int colCount ,int frontCount)
{




	int yThreads = 128;
	int yBlocks = (rowCount -1)/ yThreads +1;

	int* dBaseSum;
	int* dASD;


	CudaSafeCall( hipMalloc( (void**) &dBaseSum,sizeof(int) * colCount * rowCount   ));
	CudaSafeCall(hipMemcpy (dBaseSum, dData, sizeof(_teamReward) * colCount *rowCount, hipMemcpyDeviceToDevice));


	int blocksPerGrid_x = (colCount > 40) ? 40 : colCount;

	ColumnSum(dBaseSum,rowCount,colCount);

	// >>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>
//	int data[rowCount*colCount];
//	CudaSafeCall(hipMemcpy (data, dData, sizeof(int) * rowCount * colCount, hipMemcpyDeviceToHost));
//
//	for(int i=0;i < rowCount; i++)
//	{
//		for(int j = 0; j < colCount;j++)
//		{
//			printf(" %d", data[j+i *colCount]);
//		}
//		printf("\n");
//	}
//
//	int distSum[colCount];
//	CudaSafeCall(hipMemcpy (distSum, dBaseSum, sizeof(int) * colCount, hipMemcpyDeviceToHost));
//	printf("DataSUM\n");
//	for(int i = 0; i < colCount; i++)
//	{
//		printf(" %d ", distSum[i]);
//	}
//	printf("\n");
//
//	int* dFrontSum;

	// <<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<
	int* dWorking;
	CudaSafeCall( hipMalloc( (void**) &dWorking,sizeof(int) * colCount * rowCount   ));
	CudaSafeCall(hipMemcpy (dWorking, dData, sizeof(_teamReward) * colCount *rowCount, hipMemcpyDeviceToDevice));

	int threads = 16;
	kColumnNormalize<<<dim3((colCount-1)/threads+1,(rowCount-1)/threads+1),dim3(threads,threads),threads>>>(dWorking,dBaseSum,rowCount,colCount,dScores);
	CudaCheckError();


	for(int offset = 0; offset+blocksPerGrid_x <= colCount; offset += blocksPerGrid_x )
	{
	//	printf("adsafghgh \n");
		FrontFilter<<<dim3(blocksPerGrid_x,yBlocks),dim3(1,yThreads)>>>(dScores,dFront,0,(float)6.44,colCount, rowCount, dScores, offset);
		CudaCheckError();
	}

	// >>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>
//	float frontSum[colCount*rowCount];
//		CudaSafeCall(hipMemcpy (frontSum, dScores, sizeof(float) * colCount * rowCount, hipMemcpyDeviceToHost));
//		printf("FORNTSUM\n");
//		for(int i = 0; i < rowCount; i++)
//		{
//			for(int j = 0; j < colCount; j++)
//					{
//			printf(" %0.2f ", frontSum[j + i*colCount ]);
//		}
//		printf("\n");
//		}

	// <<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<


	RowSum(dScores,rowCount,colCount);
//	for(int _rowsLeft=rowCount; _rowsLeft >1;_rowsLeft = (_rowsLeft -1) / yThreads+1)
//	{
//		printf("rowCount: %d   %d %d\n",rowCount, colCount,yBlocks);
//		int elementsRemaining = colCount;
//		for(int offset = 0; offset+blocksPerGrid_x <= colCount; offset += blocksPerGrid_x )
//		{
//			printf("asdasDasd\n");
//			kColumnSum <<<dim3(blocksPerGrid_x,yBlocks),dim3(1,yThreads)>>>(dFrontSum,colCount, rowCount, dFrontSum,offset);
//			CudaCheckMsg("Kernel execution failed");//?? WTF???A
//		}
//
//	}

	// >>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>
	//int frontSum[colCount];
//	CudaSafeCall(hipMemcpy (frontSum, dScores, sizeof(float) * colCount * rowCount, hipMemcpyDeviceToHost));
//		printf("FORNTSUM\n");
//		for(int j = 0; j < rowCount; j++)
//		{
//			for(int i = 0; i < colCount; i++)
//				printf(" %0.2f ", frontSum[i+j*colCount]);
//			printf("\n");
//		}
//		printf("\n");

	// <<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<

//	int xThreads = 128;
//	//float* dScore;
//	//CudaSafeCall( hipMalloc( (void**) &dScore,sizeof(float) *colCount   ));
//	kElementDivide<<<dim3((colCount-1)/xThreads+1,1),dim3(xThreads)>>>(dFrontSum,dBaseSum,colCount,dScores);
//	CudaCheckMsg("Kernel execution failed");

//
//
//	CudaSafeCall(hipFree( dDistSum));
//	CudaSafeCall(hipFree( dFrontSum));
//	CudaSafeCall(hipFree( dScore));
	hipFree(dWorking);
	hipFree(dBaseSum);
}


__host__ void calcScores_F(_teamReward* dData, bool* dFront, float* dScores, int rowCount, int colCount ,int frontCount)
{
	int state = STATE_FRONT_TOO_BIG;

	int yThreads = 128;
	int yBlocks = (rowCount -1)/ yThreads +1;

	int* dBaseSum;
	int* dASD;


	CudaSafeCall( hipMalloc( (void**) &dBaseSum,sizeof(int) * colCount * rowCount   ));
	CudaSafeCall(hipMemcpy (dBaseSum, dData, sizeof(int) * colCount *rowCount, hipMemcpyDeviceToDevice));


	int blocksPerGrid_x = (colCount > 40) ? 40 : colCount;
	if(state == STATE_FRONT_TOO_BIG)
	{
	for(int offset = 0; offset+blocksPerGrid_x <= colCount; offset += blocksPerGrid_x )
		{
			FrontFilter<<<dim3(blocksPerGrid_x,yBlocks),dim3(1,yThreads)>>>(dBaseSum,dFront,1,0,colCount, rowCount, dBaseSum, offset);
			CudaCheckError();
		}
	}
	ColumnSum(dBaseSum,rowCount,colCount);



	// >>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>
//	int data[rowCount*colCount];
//	CudaSafeCall(hipMemcpy (data, dData, sizeof(int) * rowCount * colCount, hipMemcpyDeviceToHost));
//
//	for(int i=0;i < rowCount; i++)
//	{
//		for(int j = 0; j < colCount;j++)
//		{
//			printf(" %d", data[j+i *colCount]);
//		}
//		printf("\n");
//	}
//
//	int distSum[colCount];
//	CudaSafeCall(hipMemcpy (distSum, dBaseSum, sizeof(int) * colCount, hipMemcpyDeviceToHost));
//	printf("DataSUM\n");
//	for(int i = 0; i < colCount; i++)
//	{
//		printf(" %d ", distSum[i]);
//	}
//	printf("\n");
//
//	int* dFrontSum;

	// <<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<
	int* dWorking;
	CudaSafeCall( hipMalloc( (void**) &dWorking,sizeof(int) * colCount * rowCount   ));
	CudaSafeCall(hipMemcpy (dWorking, dData, sizeof(int) * colCount *rowCount, hipMemcpyDeviceToDevice));

	int threads = 16;
	kColumnNormalize<<<dim3((colCount-1)/threads+1,(rowCount-1)/threads+1),dim3(threads,threads),threads>>>(dWorking,dBaseSum,rowCount,colCount,dScores);
	CudaCheckError();
	for(int offset = 0; offset+blocksPerGrid_x <= colCount; offset += blocksPerGrid_x )
	{
		FrontFilter<<<dim3(blocksPerGrid_x,yBlocks),dim3(1,yThreads)>>>(dScores,dFront,STATE_FRONT_TOO_BIG==state,(float)6.44,colCount, rowCount, dScores, offset);
		CudaCheckError();
	}

	// >>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>
//	float frontSum[colCount*rowCount];
//		CudaSafeCall(hipMemcpy (frontSum, dScores, sizeof(float) * colCount * rowCount, hipMemcpyDeviceToHost));
//		printf("FORNTSUM\n");
//		for(int i = 0; i < rowCount; i++)
//		{
//			for(int j = 0; j < colCount; j++)
//					{
//			printf(" %0.2f ", frontSum[j + i*colCount ]);
//		}
//		printf("\n");
//		}

	// <<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<

	RowSum(dScores,rowCount,colCount);

	// >>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>
	//int frontSum[colCount];
//	CudaSafeCall(hipMemcpy (frontSum, dScores, sizeof(float) * colCount * rowCount, hipMemcpyDeviceToHost));
//		printf("FORNTSUM\n");
//		for(int j = 0; j < rowCount; j++)
//		{
//			for(int i = 0; i < colCount; i++)
//				printf(" %0.2f ", frontSum[i+j*colCount]);
//			printf("\n");
//		}
//		printf("\n");

	// <<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<
	hipFree(dWorking);
	hipFree(dBaseSum);
}


__host__ int findFront( _teamReward* dData,int rowCount, int colCount, bool* dFront , int gapSize)
{


	bool* dTmpDistDom;
	bool* dTmpDistEqual;
	CudaSafeCall( hipMalloc( (void**) &dTmpDistDom,sizeof(bool) * rowCount *rowCount *3));
	CudaSafeCall( hipMalloc( (void**) &dTmpDistEqual,sizeof(bool) *rowCount * rowCount *3 ));


	int* dTMP;
	CudaSafeCall( hipMalloc( (void**) &dTMP,sizeof(int) *rowCount * rowCount));
	CudaCheckError();
	int hTMP[rowCount*rowCount];
	CudaSafeCall(hipMemcpy (hTMP, dTMP,sizeof(int) * rowCount*rowCount , hipMemcpyDeviceToHost));
//	_teamReward tmp[rowCount*colCount];
//	CudaSafeCall(hipMemcpy (tmp, dData,sizeof(_teamReward) * rowCount*colCount , hipMemcpyDeviceToHost));
	CudaCheckError();
//	printf(" %d %p  %d  %d  %p  %p   %p\n",rowCount,dData,colCount,rowCount,dTmpDistDom,dTmpDistEqual,dTMP);

	isDominated<<<dim3(rowCount,rowCount),dim3(128)>>>(dData,colCount,rowCount,dTmpDistDom,dTmpDistEqual, dTMP);
	CudaCheckError();
	//	CudaSafeCall(hipMemcpy (hTMP, dTMP,sizeof(int) * rowCount*rowCount , hipMemcpyDeviceToHost));
//
//	printf("##$$##$$##$$\n");
//	for(int i=0;i < rowCount;i++)
//	{
//		for(int j=0;j < rowCount;j++)
//			{
//				printf(" %d", hTMP[i*rowCount+j]);
//			}
//		printf("\n");
//	}
//

	FindParetoFront(dTmpDistDom,dFront,rowCount);
	bool front[rowCount];
	CudaSafeCall(hipMemcpy (front, dFront,sizeof(bool) * rowCount , hipMemcpyDeviceToHost));

	int frontCount =0;
	for(int i=0; i< rowCount;i++)
		{
			if(front[i])
			{
				frontCount++;
			}
		}

	// >>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>.
	printf("selPoints F: ");
	for(int i=0; i< rowCount;i++)
	{
		if(front[i])
		{
			printf(" %d", i);
		//	frontCount++;
		}
	}
	printf("\nselPoints D: ");
	for(int i=0; i< rowCount;i++)
	{
		if(!front[i])
			printf(" %d", i);
	}
	printf("\n");
//	// <<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<
	 hipFree( dTmpDistDom);
	 hipFree( dTmpDistEqual);

	 hipFree(dTMP);


		return frontCount;
}
__host__ void SelectTeamsGPU(int rowCount, int colCount, _teamReward* data, int gapSize, thrust::host_vector<_teamReward>* hToDel )
{
	printf("SELECT TEAMS\n");
	_teamReward* dData;
	CudaSafeCall( hipMalloc( (void**) &dData,sizeof(_teamReward) * rowCount*colCount     ));
	CudaSafeCall(hipMemcpy (dData,data, sizeof(_teamReward) * rowCount*colCount   , hipMemcpyHostToDevice));
	CudaCheckError();

	bool* dFront;
	CudaSafeCall( hipMalloc( (void**) &dFront,sizeof(bool) * rowCount ));

	int frontCount = findFront(dData,rowCount,colCount,dFront, gapSize);

	printf(" Keep = Rowcount:%d - gapSize:%d", rowCount, gapSize);
	int keep = rowCount - gapSize;
	if(frontCount == keep)
		{
			printf(" F == SAME\n");

			bool front[rowCount];
			CudaSafeCall(hipMemcpy (front,dFront, sizeof(bool) * rowCount   , hipMemcpyDeviceToHost));
			hToDel->clear();
			for(int i=0;i<rowCount;i++)
			{
				if(!front[i])
				{
					hToDel->push_back(i);
				}
			}
		//	frontToVector(front,rowSize,hToDel);
		}
		else{

			float* dScores;
			CudaSafeCall( hipMalloc( (void**) &dScores,sizeof(float) * rowCount*colCount ));

			if (frontCount < keep)
			{
				printf(" F == TOO SMALL\n");
				calcScores_D(dData,dFront, dScores,rowCount,colCount,frontCount);

				bool front[rowCount];
				CudaSafeCall(hipMemcpy (front, dFront,sizeof(bool) * rowCount , hipMemcpyDeviceToHost));

				float scores[rowCount*colCount];
				CudaSafeCall(hipMemcpy (scores, dScores,sizeof(float) * rowCount * colCount , hipMemcpyDeviceToHost));
				hToDel->clear();
				PTeamSelectParetoSerial(frontCount < keep, front,scores, rowCount,colCount, 0, gapSize,hToDel);
			}else{
				printf(" F == TOO BIG\n");
				calcScores_F(dData,dFront, dScores,rowCount,colCount,frontCount);

				bool front[rowCount];
				CudaSafeCall(hipMemcpy (front, dFront,sizeof(bool) * rowCount , hipMemcpyDeviceToHost));

				float scores[rowCount*colCount];
				CudaSafeCall(hipMemcpy (scores, dScores,sizeof(float) * rowCount * colCount , hipMemcpyDeviceToHost));
				hToDel->clear();
				for(int i=0;i < rowCount;i++)
				{
					if(!front[i]){
						hToDel->push_back(i);
					}
				}
printf("###### , V:%d   = gap:%d - ( rowCount:%d - frontFoucnt:%d)\n",gapSize-(rowCount-frontCount),gapSize,rowCount,frontCount);
				PTeamSelectParetoSerial(frontCount < keep, front,scores, rowCount,colCount, 0, gapSize-(rowCount-frontCount),hToDel);


				//PselectParetoSerial(frontCount < keep, front,scores, rowCount, 0, gapSize,hToDel);
			}
			hipFree(dScores);
		}
		hipFree(dData);
		hipFree(dFront);
		printf("TEAM SELECT END\n");
}

__host__ void SelectPointsGPU(int rowCount, int colCount, _teamReward* data, int gapSize, thrust::host_vector<_teamReward>* hToDel )
{

	struct timeval tv_start;
	struct timeval tv_end;
	struct timezone tz;
	long timerDist =0;

	int vecCount = colCount*colCount;
	_teamReward* dDist;

	//for(int i=0;i<20; i++) { data[i*rowCount] = i%10;}
//	printf(" BALLLLS\n");
//	for(int i =0; i < colCount;i++)
//	{
//		for(int j=0;j < rowCount;j++)
//		{
//			printf("%d,", data[i*rowCount+j]);
//		}
//	printf("\n");
//	}
//
//	printf("%d = %d * %d * %d\n" ,vecCount*rowCount , colCount,colCount,rowCount  );

	//pivot
	_teamReward pivotData[rowCount*colCount];
	for(int i=0; i<rowCount ;i++)
	{
	  for(int j=0; j < colCount;j++)
	  {
	     pivotData[j*rowCount+i] =  data[i*colCount+j];
	  }
	}


	CudaSafeCall( hipMalloc( (void**) &dDist,sizeof(_teamReward) * vecCount*rowCount ));
//	printf("@Point FindFront Row:%d Col:%d \n", rowCount,vecCount);
	bool* dFront;
	CudaSafeCall( hipMalloc( (void**) &dFront,sizeof(bool) * rowCount ));

	_teamReward* dData;
	CudaSafeCall( hipMalloc( (void**) &dData,sizeof(_teamReward) * rowCount*colCount     ));
	CudaSafeCall(hipMemcpy (dData,pivotData, sizeof(_teamReward) * rowCount*colCount   , hipMemcpyHostToDevice));

	 gettimeofday(&tv_start, &tz);
	 GetDist(dData,dDist,colCount,rowCount);
	 gettimeofday(&tv_end, &tz);
	 timerDist = Diff2(tv_start,tv_end);



 //   printf("PSTAT %ld\n",timerDist);
	//GetDistTex(data,dDist,rowCount,colCount);
//	kCalcDist<<<dim3(rowCount,1),dim3(512)>>>(dData, dDist, colCount, rowCount);
	CudaCheckError();

//	//	>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>
//	_teamReward* dist_out = (_teamReward*) malloc(sizeof(_teamReward) * vecCount*rowCount);
//	CudaSafeCall(hipMemcpy (dist_out,dDist, sizeof(_teamReward) *  vecCount*rowCount/*teamCount*teamCount*pointCount*/ , hipMemcpyDeviceToHost));
//	printf("selPoints distinctions\n");
//	for(int i = 0; i < rowCount;i++)
//	{
//		for(int j = 0; j < vecCount;j++)
//		{
//			printf("%d", dist_out[j+ i*vecCount]);
//			if(j%colCount == colCount-1){printf(" ");}
//		}
//		printf("\n");
//	}
//	printf("\n");
//
//	// <<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<


//		printf("Point FindFront Row:%d Col:%d \n", rowCount,vecCount);
	int frontCount = findFront(dDist,rowCount,vecCount,dFront, gapSize);
	int keep = rowCount - gapSize;
//	printf("GAPSTATS RowCount:%d GapSize:%d  Keep:%d   FrontCount:%d\n", rowCount,gapSize, keep , frontCount);

//	printf("T:%d G:%d K:%d F:%d\n", rowCount,gapSize,keep,frontCount);
	if(frontCount == keep)
	{
//		printf(" F == SAME\n");

		bool front[rowCount];
		CudaSafeCall(hipMemcpy (front,dFront, sizeof(bool) * rowCount   , hipMemcpyDeviceToHost));
		hToDel->clear();
		for(int i=0;i<rowCount;i++)
		{
			if(!front[i])
			{
				hToDel->push_back(i);
			}
		}
	//	frontToVector(front,rowSize,hToDel);
	}
	else{

		float* dScores;
		CudaSafeCall( hipMalloc( (void**) &dScores,sizeof(float) * rowCount*colCount ));

		if (frontCount < keep)
		{
//			printf(" F == TOO SMALL\n");
			calcScores_D(dData,dFront, dScores,rowCount,colCount,frontCount);

			bool front[rowCount];
			CudaSafeCall(hipMemcpy (front, dFront,sizeof(bool) * rowCount , hipMemcpyDeviceToHost));

			float scores[rowCount*colCount];
			CudaSafeCall(hipMemcpy (scores, dScores,sizeof(float) * rowCount * colCount , hipMemcpyDeviceToHost));
			hToDel->clear();
			PointSelectParetoSerial(frontCount < keep, front,scores, rowCount,colCount, 0, gapSize,hToDel);
		}else{
//			printf(" F == TOO BIG\n");
			calcScores_F(dData,dFront, dScores,rowCount,colCount,frontCount);

			bool front[rowCount];
			CudaSafeCall(hipMemcpy (front, dFront,sizeof(bool) * rowCount , hipMemcpyDeviceToHost));

			float scores[rowCount*colCount];
			CudaSafeCall(hipMemcpy (scores, dScores,sizeof(float) * rowCount * colCount , hipMemcpyDeviceToHost));
			hToDel->clear();
			for(int i=0;i < rowCount;i++)
			{
				if(!front[i]){
					hToDel->push_back(i);
				}
			}
//			printf("HTP SIZE: %d\n", hToDel->size());
			PointSelectParetoSerial(frontCount < keep, front,scores, rowCount,colCount, 0, gapSize-(rowCount-frontCount),hToDel);


			//PselectParetoSerial(frontCount < keep, front,scores, rowCount, 0, gapSize,hToDel);
		}
		hipFree(dScores);
	}
	hipFree(dData);
	hipFree(dFront);
	hipFree(dDist);

//	printf(" Leave: PSel\n");
}






